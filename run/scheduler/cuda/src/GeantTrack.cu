#include "hip/hip_runtime.h"

#include "GeantCudaUtils.h"
#include "backend/cuda/Interface.h"

#include "CoprocessorBrokerKernel.h"
#include "GeantTaskData.h"
#include "GeantTrack.h"

namespace geant {

inline namespace cuda {
template void MakeInstanceArrayAt(GeantTaskData *addr, size_t nElements, size_t sizeOf, size_t, int, GeantPropagator *);

template void MakeInstanceAt(GeantTrack_v *addr, unsigned int, int);

__global__ void Clear(GeantTrack_v *tracks) { tracks->Clear(); }

int Clear_gpu(vecgeom::cxx::DevicePtr<geant::cuda::GeantTrack_v> &tracks, int blocksPerGrid, int threadsPerBlock,
              hipStream_t stream)
{
  Clear<<<blocksPerGrid, threadsPerBlock, 0, stream>>>(tracks);
  GEANT_CUDA_ERROR(hipGetLastError());
  return 1;
}

} // cuda
} // Geant

namespace vecgeom {
namespace cxx {
template void DevicePtr<geant::cuda::GeantConfig>::Construct() const;
template size_t DevicePtr<geant::cuda::GeantConfig>::SizeOf();
template void DevicePtr<geant::cuda::GeantPropagator>::Construct(int) const;
template size_t DevicePtr<geant::cuda::GeantPropagator>::SizeOf();
template size_t DevicePtr<geant::cuda::GeantTaskData>::SizeOf();
template size_t DevicePtr<geant::cuda::GeantTrack_v>::SizeOf();
} // cxx
} // vecgeom
