#include "RngBenchmarker.h"
#include "RngBenchmarker_gpu.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "base/Global.h"
#include "base/Stopwatch.h"
using vecgeom::Stopwatch;

namespace vecphys {

void RngBenchmarker::RunCuda()
{
  int nDevice;
  bool cudaEnabled = false;

  hipGetDeviceCount(&nDevice);
  if(nDevice > 0) {
    hipDeviceReset();
    cudaEnabled = true;
  }
  else {
    printf("Waning: No Cuda Capable Device ...\n");
  }

  //cuda event timing
  hipEvent_t start;
  hipEvent_t stop;

  hipEventCreate (&start);
  hipEventCreate (&stop);

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =   26;
  int theNThreads =  192;

  //1. MRG32k3a:

  MRG32k3a<ScalarBackend> *mrg32k2a = new MRG32k3a<ScalarBackend>();
  MRG32k3a_t<ScalarBackend>* statesMRG32k3a_d = 0; 
  hipMalloc((void**)&statesMRG32k3a_d, theNBlocks*theNThreads*sizeof(MRG32k3a_t<ScalarBackend>));
  mrg32k2a->Initialize(statesMRG32k3a_d, theNBlocks, theNThreads);

  //2. Threefry:
  Threefry<ScalarBackend> *threefry = new Threefry<ScalarBackend>();
  Threefry_t<ScalarBackend>* statesThreefry_d = 0; 
  hipMalloc((void**)&statesThreefry_d, theNBlocks*theNThreads*sizeof(Threefry_t<ScalarBackend>));
  threefry->Initialize(statesThreefry_d, theNBlocks, theNThreads);

  //Philox:
  Philox<ScalarBackend> *philox = new Philox<ScalarBackend>();
  Philox_t<ScalarBackend>* statesPhilox_d = 0; 
  hipMalloc((void**)&statesPhilox_d, theNBlocks*theNThreads*sizeof(Philox_t<ScalarBackend>));
  philox->Initialize(statesPhilox_d, theNBlocks, theNThreads);

  //4 hiprandStateMRG32k3a
  hiprandStateMRG32k3a* devStatesMRG32k3a = 0;
  hipMalloc(&devStatesMRG32k3a,theNBlocks*theNThreads*sizeof(hiprandStateMRG32k3a));
  curand_setup_gpu(devStatesMRG32k3a, time(NULL), theNBlocks, theNThreads);

  //4 curandStatePhilox
  hiprandStatePhilox4_32_10_t* devStatesPhilox = 0;
  hipMalloc(&devStatesPhilox,theNBlocks*theNThreads*sizeof(hiprandStateMRG32k3a));
  curand_setup_gpu(devStatesPhilox, time(NULL), theNBlocks, theNThreads);

  //return values for varification
  double *result_h;
  double *result_c;
  double *result_d;

  result_h = (double*) calloc(theNBlocks*theNThreads, sizeof(double));
  result_c = (double*) calloc(theNBlocks*theNThreads, sizeof(double));
  hipMalloc((void**)&result_d,theNBlocks*theNThreads*sizeof(double));

  float elapsedEventTime[kNumberRng +2];
  float elapsedTotalTime[kNumberRng +2];
  double rngTotal[kNumberRng +2];
  double rngEvent[kNumberRng +2];

  for (int k = 0; k < kNumberRng + 2 ; ++k) {
    elapsedTotalTime[k] = 0.;
    rngTotal[k] = 0.;
  }

  for (unsigned r = 0; r < fRepetition; ++r) {

    for (int k = 0; k < kNumberRng + 2; ++k) {
      elapsedEventTime[k] = 0.0;
      rngEvent[k] = 0.0;

      hipMemset(result_d,0,theNBlocks*theNThreads*sizeof(double));
  
      if(cudaEnabled) {
        hipEventRecord (start,0);

        //call CUDA kernel
	if(k == 0) {
	  CudaMRG32k3a(statesMRG32k3a_d, result_d, fNSample, theNBlocks, theNThreads);
        }
	if(k == 1) {
     	  CudaThreefry(statesThreefry_d, result_d, fNSample, theNBlocks, theNThreads);
	}
	if(k == 2) {
	  CudaPhilox(statesPhilox_d, result_d, fNSample, theNBlocks, theNThreads);
	}

	if(k == 3) {
          CurandMRG32k3a(devStatesMRG32k3a,result_d,fNSample,theNBlocks,theNThreads);
	}

	if(k == 4) {
          CurandPhilox(devStatesPhilox,result_d,fNSample,theNBlocks,theNThreads);
	}

        hipEventRecord (stop,0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedEventTime[k],start,stop);

        //copy the result for varification
        hipMemcpy(result_h,result_d,theNBlocks*theNThreads*sizeof(double),
                hipMemcpyDeviceToHost);
         
        for(int i = 0 ; i < theNBlocks*theNThreads ; ++i) rngEvent[k] += result_h[k]; 
      }
      elapsedTotalTime[k] += elapsedEventTime[k]/1000.; //ms to second
      rngTotal[k] += rngEvent[k]; //ms to second
    }
  }

  for (int k = 0; k < kNumberRng + 2; ++k) {
    if(k < kNumberRng) {
      printf(" %s  CudaBackend   Total time = %7.4f sec CheckSum = %g\n", 
   	     RngName[k], elapsedTotalTime[k], rngTotal[k]);
    }
    if(k== kNumberRng) {
      printf("%s Nvidia   Total time = %7.4f sec CheckSum = %g\n", 
   	     "CurandMRG32k3a", elapsedTotalTime[k], rngTotal[k]);
    }
    if(k== kNumberRng+1) {
      printf("%s Nvidia   Total time = %7.4f sec CheckSum = %g\n", 
   	     "CurandPhilox  ", elapsedTotalTime[k], rngTotal[k]);
    }
  }

  //clean up: destory cuda event and free memory on device and host
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(statesMRG32k3a_d);
  hipFree(statesThreefry_d);
  hipFree(statesPhilox_d);
  hipFree(devStatesMRG32k3a);
  hipFree(devStatesPhilox);

  hipFree(result_d);
  free(result_h);
  free(result_c);

  delete mrg32k2a;
  delete threefry;
  delete philox;
}

} // end of vecphys namespace

