#include "trackingTest2_kernel.h"
#include "gxtracking_kernel.h"
#include "sort_kernel.h"

#include "GXTrackLiason.h"

//#include "gxtracking_kernel.cu"
//cuda/kernel/src/trackingTest2_kernel.cu

int electron_gpu(hiprandState* devStates,
                 size_t nSteps,
                 size_t nElectrons,
                 GXTrack *track, GXTrack * /* altTrack */,
                 int *logVolumeIndices,
                 int *physVolumeIndices,
                 GXTrack *secondaries, int *secStackSize,
                 
                 int *scratch,

                 GPGeomManager *geomManager,
                 GXFieldMap *magMap,
                 GPPhysicsTable *physicsTable,
                 GPPhysics2DVector *seltzerBergerTable,

                 int nBlocks, int nThreads,
                 hipStream_t stream)
{
   int *secOffset = &(scratch[0]);

   elec_gpu(devStates, track,
            logVolumeIndices, physVolumeIndices,
            nElectrons, 
            geomManager, magMap, 
            physicsTable, seltzerBergerTable,
            secondaries, secStackSize, secOffset,
            nSteps, 
            0 /* runType */,
            nBlocks, nThreads, stream);
   return 0;
}

int electron_multistage_gpu(hiprandState* devStates,
                            size_t nSteps,
                            size_t nElectrons,
                            GXTrack *track, GXTrack *altTrack,
                            int *logVolumeIndices,
                            int *physVolumeIndices,
                            GXTrack *secondaries, int *secStackSize,

                            int *scratch, // array of 10.
                             
                            GPGeomManager *geomManager,
                            GXFieldMap *magMap,
                            GPPhysicsTable *physicsTable,
                            GPPhysics2DVector *seltzerBergerTable,
                            
                            int nBlocks, int nThreads,
                            hipStream_t stream)
{
   int *nbrem = &(scratch[0]);
   int *nioni = &(scratch[1]);
   int *stackSize_brem = &(scratch[2]);
   int *stackSize_ioni = &(scratch[3]);
   int *secOffset = &(scratch[4]);

   GXTrackLiason *liason_d;
   hipMalloc((void**)&liason_d, nElectrons*sizeof(GXTrackLiason));

   elec_GPIL_gpu(devStates, track, 
                 liason_d, 
                 logVolumeIndices, physVolumeIndices,
                 nElectrons,
                 geomManager, magMap, 
                 physicsTable, seltzerBergerTable,
                 secondaries, secStackSize, secOffset, 
                 nSteps,
                 0 /* runType */,
                 nBlocks, nThreads, stream);
   
   // //atomic counter for the last array position of physics processes
   // hipDeviceSynchronize();
   
   count_by_process_gpu(nElectrons, track,
                        nbrem, nioni, nBlocks, nThreads, stream);
   
   //   hipMemcpyAsync(&nbrem_h,nbrem_d,sizeof(G4int),hipMemcpyDeviceToHost,stream);
   //   hipMemcpyAsync(&nioni_h,nioni_d,sizeof(G4int),hipMemcpyDeviceToHost,stream);
   
   // This copies the track from track to altTrack
   sort_by_process_gpu(nElectrons, track, altTrack,
                       nbrem, stackSize_brem,
                       nioni, stackSize_ioni,
                       nBlocks, nThreads, stream);
   
   // hipDeviceSynchronize();
   
   elec_doit_gpu(devStates, altTrack, 
                 liason_d, 
                 nElectrons, 
                 geomManager, magMap, 
                 physicsTable,seltzerBergerTable,
                 secondaries, secStackSize, secOffset,
                 nSteps, 
                 0 /* runType */,
                 nBlocks, nThreads, stream);

   hipFree(liason_d);
   return 1; // The real data is in altTrack
}