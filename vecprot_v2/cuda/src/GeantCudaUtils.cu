// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"
#include "GeantPropagator.h"
#include "GeantTrack.h"
#include "TGeoShape.h"
#include "globals.h"

namespace Geant {
inline namespace cuda {

void CoprocessorBrokerInitConstant()
{
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(gPropagator_fBmag), &(gPropagator->fBmag), sizeof(double), size_t(0), hipMemcpyHostToDevice));

  double tolerance = TGeoShape::Tolerance();
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(device_constant::gTolerance), &(tolerance), sizeof(double), size_t(0), hipMemcpyHostToDevice));
}

} // cuda
} // Geant
