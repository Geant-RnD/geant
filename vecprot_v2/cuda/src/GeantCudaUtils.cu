// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"

namespace Geant {
inline namespace cuda {

void CoprocessorBrokerInitConstant() {
   GEANT_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(gPropagator_fBmag), &(gPropagator->fBmag), sizeof(double), size_t(0), hipMemcpyHostToDevice) );

   double tolerance = TGeoShape::Tolerance();
   GEANT_CUDA_ERROR( hipMemcpyToSymbol(HIP_SYMBOL(device_constant::gTolerance), &(tolerance), sizeof(double), size_t(0), hipMemcpyHostToDevice) );
}

} // cuda
} // Geant
