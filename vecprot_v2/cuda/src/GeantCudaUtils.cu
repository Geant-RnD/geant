// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"
#include "GeantPropagator.h"
#include "GeantTrack.h"
#ifndef USE_VECGEOM_NAVIGATOR
#include "TGeoShape.h"
#endif
#include "globals.h"

namespace Geant {
inline namespace cuda {

void CoprocessorBrokerInitConstant()
{
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(gPropagator_fBmag), &(gPropagator->fBmag), sizeof(double), size_t(0), hipMemcpyHostToDevice));

#ifndef USE_VECGEOM_NAVIGATOR
  double tolerance = TGeoShape::Tolerance();
#else
  double tolerance = 1e-7;
#endif
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(device_constant::gTolerance), &(tolerance), sizeof(double), size_t(0), hipMemcpyHostToDevice));
}

} // cuda
} // Geant
