// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"

namespace Geant {
inline namespace cuda {

void Fatal(const char *location, const char *va_(fmt), ...)
{
   // Use this function in case of a fatal error. It will abort the program.

   va_list ap;
   va_start(ap,va_(fmt));
   printf("Fatal in <%s>:",location);
   printf(va_(fmt), ap);
   printf("\n");
   va_end(ap);
#ifdef GEANT_CUDA_DEVICE_BUILD
   //hipDeviceReset();
   hipDeviceReset();
#else
   exit( EXIT_FAILURE );
#endif
}

} // cxx
} // Geant
