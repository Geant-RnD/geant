// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"
#include "GeantPropagator.h"
#include "GeantTrack.h"
#include "GeantConfig.h"
#include "globals.h"

namespace Geant {
inline namespace cuda {

void CoprocessorBrokerInitConstant()
{
  double tolerance = 1e-7;
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(device_constant::gTolerance), &(tolerance), sizeof(double), size_t(0), hipMemcpyHostToDevice));
}

} // cuda
} // Geant
