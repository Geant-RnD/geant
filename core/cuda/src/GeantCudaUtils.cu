// See also GeantCudaUtils.cxx

#include "GeantCudaUtils.h"
#include "GeantPropagator.h"
#include "GeantTrack.h"
#include "GeantConfig.h"
#include "globals.h"

namespace geant {
inline namespace cuda {

void CoprocessorBrokerInitConstant()
{
  double tolerance = 1e-7;
  GEANT_CUDA_ERROR(
      hipMemcpyToSymbol(HIP_SYMBOL(device_constant::gTolerance), &(tolerance), sizeof(double), size_t(0), hipMemcpyHostToDevice));
}

} // cuda
} // Geant
