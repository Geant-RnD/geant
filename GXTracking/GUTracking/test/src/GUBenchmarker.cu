#include "hip/hip_runtime.h"
#include "GUBenchmarker.h"

#include "base/Stopwatch.h"
#include "backend/cuda/Backend.h"

#include "GUAliasSampler.h"
#include "GUComptonKleinNishina.h"
#include "GUTrackHandler.h"

#include "GUCurand.h"

namespace vecphys {
inline namespace cuda {

__global__
void BenchmarkCudaKernel(Random_t* devStates,
                         GUAliasTable* table,
			 int nTrackSize, 
                         GUTrack* itrack, 
			 int* targetElements, 
			 GUTrack* otrack)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GUAliasSampler sampler(devStates,tid,10,1.,1001.,100,100,table);
  GUComptonKleinNishina model(devStates,tid,&sampler);

  while (tid < nTrackSize) {
    model.Interact<kCuda>(itrack[tid],targetElements[tid],otrack[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

} // end namespace cuda

void GUBenchmarker::RunCuda()
{
  hipDeviceReset();

  //set 1024 megabytes on the heap (global mememory) 
  cudaThreadSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);

  //prepare table
  GUComptonKleinNishina *model = new GUComptonKleinNishina(0,-1);
  GUAliasTable* table_h =  model->GetSampler()->GetAliasTable();

  GUAliasTable* table_d;
  hipMalloc((void**)&table_d,table_h->SizeOfTable());
  table_h->Relocate(table_d);

  GUTrack* otrack_aos = (GUTrack*) malloc(fNtracks*sizeof(GUTrack));

  int *targetElements = new int [fNtracks];
  for(int i = 0 ; i < fNtracks ; ++i) {
    targetElements[i] = i ;
  }
  
  int *targetElements_d;
  hipMalloc((void**)&targetElements_d, fNtracks*sizeof(int));
  hipMemcpy(targetElements_d, targetElements, fNtracks*sizeof(int), 
               hipMemcpyHostToDevice);

  //allocate memory for input/output tracks
  GUTrack *itrack_d;
  GUTrack *otrack_d;
  hipMalloc((void**)&itrack_d, fNtracks*sizeof(GUTrack));
  hipMalloc((void**)&otrack_d, fNtracks*sizeof(GUTrack));

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =  26;
  int theNThreads = 192;

  //prepare random engines on the device
  Random_t* randomStates = 0;
  hipMalloc(&randomStates, theNBlocks*theNThreads* sizeof(hiprandState));
  GUCurand_Init(randomStates, time(NULL), theNBlocks, theNThreads);

  Stopwatch timer;

  for (unsigned r = 0; r < fRepetitions; ++r) {

    fTrackHandler->GenerateRandomTracks(fNtracks);
    GUTrack* itrack_aos = fTrackHandler->GetAoSTracks();
    hipMemcpy(itrack_d, itrack_aos, fNtracks*sizeof(GUTrack), 
                 hipMemcpyHostToDevice);

    timer.Start();
    vecphys::cuda::BenchmarkCudaKernel<<<theNBlocks, theNThreads>>>(
				       randomStates,table_d,fNtracks,
				       itrack_d, targetElements_d,otrack_d);
    hipDeviceSynchronize();
    Precision elapsedCuda = timer.Stop();

    hipMemcpy(otrack_aos, otrack_d, fNtracks*sizeof(GUTrack), 
               hipMemcpyDeviceToHost);

    if (fVerbosity > 0) {
      printf("CUDA   Task %d >: %6.3fs\n",r,elapsedCuda);
    }

  }

  if (fVerbosity > 1) {
    for(unsigned i = 0; i < 4 ; ++i) printf(" E[%d]= %f\n",i,otrack_aos[i].E);
  }

  hipFree(randomStates);
  hipFree(itrack_d);
  hipFree(otrack_d);
  hipFree(table_d);
  hipFree(targetElements_d);

  free(table_h);
  free(targetElements);
  free(otrack_aos);
  //  delete model;
}

} // end of vecphys namespace

