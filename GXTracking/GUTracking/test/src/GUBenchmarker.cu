#include "GUBenchmarker.h"
#include "GUBenchmarker_gpu.h"
#include "GUPhysicsModelName.h"

#include "base/Stopwatch.h"
#include "backend/cuda/Backend.h"

#include "GUAliasSampler.h"
//#include "GVComptonKleinNishina.h"
#include "ComptonKleinNishina.h"
#include "ConversionBetheHeitler.h"
#include "PhotoElectronSauterGavrila.h"
#include "IonisationMoller.h"
#include "BremSeltzerBerger.h"
#include "Physics2DVector.h"
#include "GUTrackHandler.h"

#include "SamplingMethod.h"

#ifdef VECPHYS_ROOT
#include "GUHistogram.h"
#endif

#include "GUCurand.h"

namespace vecphys {

void GUBenchmarker::RunCuda()
{
  int nDevice;
  bool cudaEnabled = false;

  hipGetDeviceCount(&nDevice);
  if(nDevice > 0) {
    hipDeviceReset();
    cudaEnabled = true;
    //    printf("CUDA Enabled with %d Device(s)\n",nDevice);
  }
  else {
    printf("Waning: No Cuda Capable Device ...");
  }

  //set 1024 megabytes on the heap (global mememory) 
  //cudaThreadSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);

  //cuda event timing
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

#ifdef VECPHYS_ROOT
  GUHistogram* histogram = new GUHistogram("cuda.root", fMaxP);
#endif

  int *targetElements = new int [fNtracks];
  int *targetElements_d;

  //prepare table - this step may be move to the physics list later
  GUAliasTableManager** tableM_h = 
    (GUAliasTableManager**) malloc(kNumberPhysicsModel*sizeof(GUAliasTableManager*)); 

  ComptonKleinNishina *KleinNishina;
  if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == kKleinNishina) {      
    KleinNishina = new ComptonKleinNishina(0,-1);
    tableM_h[kKleinNishina] = KleinNishina->GetSampler()->GetAliasTableManager();
  }

  ConversionBetheHeitler *BetheHeitler;
  if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == kBetheHeitler) {      
    BetheHeitler = new ConversionBetheHeitler(0,-1);
    tableM_h[kBetheHeitler] = BetheHeitler->GetSampler()->GetAliasTableManager();
  }

  PhotoElectronSauterGavrila *SauterGavrila;
  if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == kSauterGavrila) {      
    SauterGavrila = new PhotoElectronSauterGavrila(0,-1);
    tableM_h[kSauterGavrila] = SauterGavrila->GetSampler()->GetAliasTableManager();
  }

  IonisationMoller *MollerBhabha;
  if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == kMollerBhabha) {      
    MollerBhabha = new IonisationMoller(0,-1);
    tableM_h[kMollerBhabha] = MollerBhabha->GetSampler()->GetAliasTableManager();
  }

  BremSeltzerBerger *SeltzerBerger;
  Physics2DVector* sbData;
  Physics2DVector* sbData_d;

  if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == kSeltzerBerger) {      
    SeltzerBerger = new BremSeltzerBerger(0,-1);
    tableM_h[kSeltzerBerger] = SeltzerBerger->GetSampler()->GetAliasTableManager();
  //SeltzerBerger data
    sbData = SeltzerBerger->GetSBData();
    hipMalloc((void**)&sbData_d,maximumZ*sizeof(Physics2DVector));
    hipMemcpy(sbData_d, sbData, maximumZ*sizeof(Physics2DVector),
	       hipMemcpyHostToDevice);
  }

  GUAliasTableManager** tableM_d;
  hipMalloc((void**)&tableM_d,kNumberPhysicsModel*sizeof(GUAliasTableManager*));

  GUAliasTableManager* temp_d[kNumberPhysicsModel];
  for(int i = 0 ; i < kNumberPhysicsModel ; ++i) {
    if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == i) {      
      hipMalloc((void**)&temp_d[i],tableM_h[i]->SizeOfManager());
      tableM_h[i]->Relocate(temp_d[i]);
    }
  }

  hipMemcpy(tableM_d,temp_d,sizeof(GUAliasTableManager*)*kNumberPhysicsModel,
             hipMemcpyHostToDevice);

  GUTrack* itrack_aos = (GUTrack*) malloc(fNtracks*sizeof(GUTrack));
  GUTrack* otrack_aos = (GUTrack*) malloc(fNtracks*sizeof(GUTrack));

  //allocate memory for input/output tracks
  GUTrack *itrack_d;
  GUTrack *otrack_d;
  hipMalloc((void**)&itrack_d, fNtracks*sizeof(GUTrack));
  hipMalloc((void**)&otrack_d, fNtracks*sizeof(GUTrack));

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =  26;
  int theNThreads = 192;

  //prepare random engines on the device
  Random_t* randomStates = 0;
  hipMalloc(&randomStates, theNBlocks*theNThreads* sizeof(hiprandState));
  GUCurand_Init(randomStates, time(NULL), theNBlocks, theNThreads);

  float elapsedTotal[kNumberPhysicsModel];
  float elapsedEventTime[kNumberPhysicsModel];

  for(int k = 0 ; k < kNumberPhysicsModel ; ++k) elapsedTotal[k] = 0.; 

  for (unsigned r = 0; r < fRepetitions; ++r) {

    PrepareTargetElements(targetElements, fNtracks);

    //H2D
    hipMalloc((void**)&targetElements_d, fNtracks*sizeof(int));
    hipMemcpy(targetElements_d, targetElements, fNtracks*sizeof(int), 
               hipMemcpyHostToDevice);

    fTrackHandler->GenerateRandomTracks(fNtracks,fMinP, fMaxP);
    GUTrack* track_aos = fTrackHandler->GetAoSTracks();

    for(int k = 0 ; k < kNumberPhysicsModel ; ++k) {
      if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == k) {     

	fTrackHandler->CopyAoSTracks(track_aos,itrack_aos);
	hipMemcpy(itrack_d, track_aos, fNtracks*sizeof(GUTrack), 
                   hipMemcpyHostToDevice);
	
	elapsedEventTime[k] = 0.0;
	
	if(cudaEnabled) {
	  hipEventRecord (start,0);
	  //call CUDA kernels
	  CudaKernelFunc[k](theNBlocks, theNThreads, randomStates,
			    tableM_d,sbData_d,fNtracks, itrack_d, 
			    targetElements_d,otrack_d,fSampleType);
	  hipEventRecord (stop,0);
	  hipEventSynchronize (stop);
	  hipEventElapsedTime (&elapsedEventTime[k],start,stop);
	}
	elapsedTotal[k] += elapsedEventTime[k]/1000.; //ms to second
	
	hipMemcpy(itrack_aos, itrack_d, fNtracks*sizeof(GUTrack), 
		   hipMemcpyDeviceToHost);
	hipMemcpy(otrack_aos, otrack_d, fNtracks*sizeof(GUTrack), 
		   hipMemcpyDeviceToHost);
	
#ifdef VECPHYS_ROOT
	histogram->RecordTime(k,elapsedEventTime[k]);
	for(int i = 0 ; i < fNtracks ; ++i) {
	  histogram->RecordHistos(k,track_aos[i].E,
				  itrack_aos[i].E,
				  itrack_aos[i].pz/itrack_aos[i].E,
				  otrack_aos[i].E,
				  otrack_aos[i].pz/otrack_aos[i].E);
	} 
#endif    
      }
    }
  }

  for(int k = 0 ; k < kNumberPhysicsModel ; ++k) {
    if(fEmModel == GUPhysicsModelIndex::kNullModel || fEmModel == k) {     
      printf("%s  CUDA   Total time of %3d reps = %7.4f sec\n",
	     GUPhysicsModelName[k], fRepetitions, elapsedTotal[k]);
    }
  }

  //clean up: destory cuda event and free memory on device and host
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(randomStates);
  hipFree(itrack_d);
  hipFree(otrack_d);
  hipFree(tableM_d);
  hipFree(targetElements_d);

  free(tableM_h);
  free(targetElements);
  free(itrack_aos);
  free(otrack_aos);

  //  delete model;
#ifdef VECPHYS_ROOT
  delete histogram;
#endif
}

} // end of vecphys namespace

