#include "GUBenchmarker.h"
#include "GUBenchmarker_gpu.h"
#include "GUPhysicsModelName.h"

#include "base/Stopwatch.h"
#include "backend/cuda/Backend.h"

#include "GUAliasSampler.h"
//#include "GVComptonKleinNishina.h"
#include "ComptonKleinNishina.h"
#include "ConversionBetheHeitler.h"
#include "PhotoElectronSauterGavrila.h"
#include "IonisationMoller.h"
#include "BremSeltzerBerger.h"
#include "Physics2DVector.h"
#include "GUTrackHandler.h"

#include "SamplingMethod.h"

#ifdef VECPHYS_ROOT
#include "GUHistogram.h"
#endif

#include "GUCurand.h"

namespace vecphys {

void GUBenchmarker::RunCuda()
{
  int nDevice;
  bool cudaEnabled = false;

  hipGetDeviceCount(&nDevice);
  if(nDevice > 0) {
    hipDeviceReset();
    cudaEnabled = true;
    printf("CUDA Enabled with %d Device(s)\n",nDevice);
  }
  else {
    printf("Waning: No Cuda Capable Device ...");
  }

  //set 1024 megabytes on the heap (global mememory) 
  //cudaThreadSetLimit(hipLimitMallocHeapSize, 1024*1024*1024);

  //cuda event timing
  hipEvent_t start, stop;
  hipEventCreate (&start);
  hipEventCreate (&stop);

#ifdef VECPHYS_ROOT
  GUHistogram* histogram = new GUHistogram("cuda.root", fMaxP);
#endif

  int *targetElements = new int [fNtracks];
  int *targetElements_d;

  //prepare table - this step may be move to the physics list later
  ComptonKleinNishina *KleinNishina = new ComptonKleinNishina(0,-1);
  ConversionBetheHeitler *BetheHeitler = new ConversionBetheHeitler(0,-1);
  PhotoElectronSauterGavrila *SauterGavrila = new PhotoElectronSauterGavrila(0,-1);
  IonisationMoller *MollerBhabha = new IonisationMoller(0,-1);
  BremSeltzerBerger *SeltzerBerger = new BremSeltzerBerger(0,-1);

  GUAliasTableManager** tableM_h = 
    (GUAliasTableManager**) malloc(kNumberPhysicsModel*sizeof(GUAliasTableManager*)); 

  tableM_h[kKleinNishina]  = KleinNishina->GetSampler()->GetAliasTableManager();
  tableM_h[kBetheHeitler]  = BetheHeitler->GetSampler()->GetAliasTableManager();
  tableM_h[kSauterGavrila] = SauterGavrila->GetSampler()->GetAliasTableManager();
  tableM_h[kMollerBhabha]  = MollerBhabha->GetSampler()->GetAliasTableManager();
  tableM_h[kSeltzerBerger] = SeltzerBerger->GetSampler()->GetAliasTableManager();

  GUAliasTableManager** tableM_d;
  hipMalloc((void**)&tableM_d,kNumberPhysicsModel*sizeof(GUAliasTableManager*));

  //  KleinNishina->GetSampler()->PrintTable();
  
  GUAliasTableManager* temp_d[kNumberPhysicsModel];
  for(int i = 0 ; i < kNumberPhysicsModel ; ++i) {
    hipMalloc((void**)&temp_d[i],tableM_h[i]->SizeOfManager());
    tableM_h[i]->Relocate(temp_d[i]);
  }

  hipMemcpy(tableM_d,temp_d,sizeof(GUAliasTableManager*)*kNumberPhysicsModel,
             hipMemcpyHostToDevice);

  //SeltzerBerger data
  Physics2DVector* sbData = SeltzerBerger->GetSBData();
  Physics2DVector* sbData_d;
  hipMalloc((void**)&sbData_d,maximumZ*sizeof(Physics2DVector));
  hipMemcpy(sbData_d, sbData, maximumZ*sizeof(Physics2DVector),
             hipMemcpyHostToDevice);

  GUTrack* itrack_aos = (GUTrack*) malloc(fNtracks*sizeof(GUTrack));
  GUTrack* otrack_aos = (GUTrack*) malloc(fNtracks*sizeof(GUTrack));

  //allocate memory for input/output tracks
  GUTrack *itrack_d;
  GUTrack *otrack_d;
  hipMalloc((void**)&itrack_d, fNtracks*sizeof(GUTrack));
  hipMalloc((void**)&otrack_d, fNtracks*sizeof(GUTrack));

  //set the default number of threads and thread blocks - should be setable
  int theNBlocks  =  26;
  int theNThreads = 192;

  //prepare random engines on the device
  Random_t* randomStates = 0;
  hipMalloc(&randomStates, theNBlocks*theNThreads* sizeof(hiprandState));
  GUCurand_Init(randomStates, time(NULL), theNBlocks, theNThreads);

  float elapsedTotal[kNumberPhysicsModel];
  float elapsedEventTime[kNumberPhysicsModel];

  for(int k = 0 ; k < kNumberPhysicsModel ; ++k) elapsedTotal[k] = 0.; 

  for (unsigned r = 0; r < fRepetitions; ++r) {

    PrepareTargetElements(targetElements, fNtracks);

    //H2D
    hipMalloc((void**)&targetElements_d, fNtracks*sizeof(int));
    hipMemcpy(targetElements_d, targetElements, fNtracks*sizeof(int), 
               hipMemcpyHostToDevice);

    fTrackHandler->GenerateRandomTracks(fNtracks,fMinP, fMaxP);
    GUTrack* track_aos = fTrackHandler->GetAoSTracks();

    for(unsigned int k = 0 ; k < kNumberPhysicsModel ; ++k) {

      fTrackHandler->CopyAoSTracks(track_aos,itrack_aos);
      hipMemcpy(itrack_d, track_aos, fNtracks*sizeof(GUTrack), 
                   hipMemcpyHostToDevice);

      elapsedEventTime[k] = 0.0;

      if(cudaEnabled) {
        hipEventRecord (start,0);
	//call CUDA kernels
        CudaKernelFunc[k](theNBlocks, theNThreads, randomStates,
                                      tableM_d,sbData_d,fNtracks, itrack_d, 
                                      targetElements_d,otrack_d,fSampleType);
        hipEventRecord (stop,0);
        hipEventSynchronize (stop);
        hipEventElapsedTime (&elapsedEventTime[k],start,stop);
      }
      elapsedTotal[k] += elapsedEventTime[k]/1000.; //ms to second

      hipMemcpy(itrack_aos, itrack_d, fNtracks*sizeof(GUTrack), 
                 hipMemcpyDeviceToHost);
      hipMemcpy(otrack_aos, otrack_d, fNtracks*sizeof(GUTrack), 
                 hipMemcpyDeviceToHost);

#ifdef VECPHYS_ROOT
      histogram->RecordTime(k,elapsedEventTime[k]);
      for(int i = 0 ; i < fNtracks ; ++i) {
        histogram->RecordHistos(k,track_aos[i].E,
	  		        itrack_aos[i].E,
			        itrack_aos[i].pz/itrack_aos[i].E,
			        otrack_aos[i].E,
			        otrack_aos[i].pz/otrack_aos[i].E);
      } 
#endif    
    }
  }

  for(int k = 0 ; k < kNumberPhysicsModel ; ++k) {
    printf("%s  CUDA   Total time of %3d reps = %8.4f sec\n",
           GUPhysicsModelName[k], fRepetitions, elapsedTotal[k]);
  }

  //clean up: destory cuda event and free memory on device and host
  hipEventDestroy(start);
  hipEventDestroy(stop);

  hipFree(randomStates);
  hipFree(itrack_d);
  hipFree(otrack_d);
  hipFree(tableM_d);
  hipFree(targetElements_d);

  free(tableM_h);
  free(targetElements);
  free(itrack_aos);
  free(otrack_aos);

  //  delete model;
#ifdef VECPHYS_ROOT
  delete histogram;
#endif
}

} // end of vecphys namespace

