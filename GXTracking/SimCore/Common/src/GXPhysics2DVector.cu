#include "hip/hip_runtime.h"
#include "GXPhysics2DVector.h"

FQUALIFIER GXPhysics2DVector::GXPhysics2DVector()
{
  for(size_t j = 0; j<numberOfYNodes; ++j) {
    yVector[j] = 0.0;
  }

  for(size_t i=0; i<numberOfXNodes; ++i) {
    xVector[i] = 0.0;
    for(size_t j=0; j<numberOfYNodes; ++j) {
      value[j][i] = 0.0;
    }
  }
};

FQUALIFIER 
G4double GXPhysics2DVector::Value(G4double x, G4double y)
{
  // no interpolation outside the table
  if(x < xVector[0]) { 
    x = xVector[0]; 
  } else if(x > xVector[numberOfXNodes - 1]) { 
    x = xVector[numberOfXNodes - 1]; 
  }
  if(y < yVector[0]) { 
    y = yVector[0]; 
  } else if(y > yVector[numberOfYNodes - 1]) { 
    y = yVector[numberOfYNodes - 1]; 
  }

  // find bins
  size_t idx = FindBinLocationX(x);
  size_t idy = FindBinLocationY(y);

  // interpolate
  //  if(useBicubic) {
  //    return BicubicInterpolation(x, y, idx, idy);
  //  } else {
  // useBicubic = false
  G4double x1 = xVector[idx];
  G4double x2 = xVector[idx+1];
  G4double y1 = yVector[idy];
  G4double y2 = yVector[idy+1];
  G4double v11= GetValue(idx,   idy);
  G4double v12= GetValue(idx+1, idy);
  G4double v21= GetValue(idx,   idy+1);
  G4double v22= GetValue(idx+1, idy+1);
  return ((y2 - y)*(v11*(x2 - x) + v12*(x - x1)) + 
	  ((y - y1)*(v21*(x2 - x) + v22*(x - x1))))/((x2 - x1)*(y2 - y1)); 
  //  }
}

FQUALIFIER
void GXPhysics2DVector::PutX(size_t idx, G4double val)
{
  xVector[idx] = val;
}

FQUALIFIER
void GXPhysics2DVector::PutY(size_t idy, G4double val)
{
  yVector[idy] = val;
}

FQUALIFIER void 
GXPhysics2DVector::PutValue(size_t idx, size_t idy, G4double val)
{
  value[idy][idx] = val;
}

FQUALIFIER
G4double GXPhysics2DVector::GetValue(size_t idx, size_t idy)
{
  return value[idy][idx];
}

FQUALIFIER 
size_t GXPhysics2DVector::FindBinLocationX(G4double z)
{
  size_t id = 0;
  if(z < xVector[1]) { 
    id = 0; 
  } 
  else if(z >= xVector[numberOfXNodes-2]) { 
    id = numberOfXNodes - 2; 
  } 
  else {
    size_t lowerBound = 0;
    size_t upperBound = numberOfXNodes - 2;

    while (lowerBound <= upperBound) {
      size_t midBin = (lowerBound + upperBound)/2;
      if( z < xVector[midBin] ) { upperBound = midBin-1; }
      else                      { lowerBound = midBin+1; }
    }
    id = upperBound;
  }
  return id;
}

FQUALIFIER 
size_t GXPhysics2DVector::FindBinLocationY(G4double z)
{
  size_t id = 0;
  if(z < yVector[1]) { 
    id = 0; 
  } 
  else if(z >= yVector[numberOfYNodes-2]) { 
    id = numberOfYNodes - 2; 
  } 
  else {
    size_t lowerBound = 0;
    size_t upperBound = numberOfYNodes - 2;

    while (lowerBound <= upperBound) {
      size_t midBin = (lowerBound + upperBound)/2;
      if( z < yVector[midBin] ) { upperBound = midBin-1; }
      else                      { lowerBound = midBin+1; }
    }
    id = upperBound;
  }
  return id;
}
