#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include "GPTypeDef.h"

#ifdef __HIPCC__
GLOBALFUNC void curand_setup_kernel(hiprandState *devStates, unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);
}

GLOBALFUNC void curand_setup_kernel(hiprandStateMRG32k3a *devStates, unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);

}

void curand_setup_gpu(hiprandState *devStates, unsigned long seed,  int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<< blocksPerGrid, threadsPerBlock >>> (devStates,seed);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "cuda_setup_kernel status = " << kstatus << "\n";

}

void curand_setup_gpu(hiprandStateMRG32k3a *devStates, unsigned long seed,  int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<< blocksPerGrid, threadsPerBlock >>> (devStates,seed);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "cuda_setup_kernel status = " << kstatus << "\n";

}
#endif

