#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include "GPTypeDef.h"
#include <stdio.h>

#ifdef __HIPCC__
GLOBALFUNC void curand_setup_kernel(hiprandState *devStates, unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);
}

GLOBALFUNC void curand_setup_kernel(hiprandStateMRG32k3a *devStates, unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);

}

bool curand_setup_gpu(hiprandState *devStates, unsigned long seed,  int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<< blocksPerGrid, threadsPerBlock >>> (devStates,seed);

  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
     {
        fprintf( stderr, "curand_setup_kernel cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
        // or 
        // return false;
     }

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "curand_setup_kernel status = " << kstatus << "\n";

  return true;
}

bool curand_setup_gpu(hiprandStateMRG32k3a *devStates, unsigned long seed,  int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<< blocksPerGrid, threadsPerBlock >>> (devStates,seed);

  hipError_t err = hipGetLastError();
  if ( hipSuccess != err )
     {
        fprintf( stderr, "curand_setup_kernel cudaCheckError() failed at %s:%i : %s\n",
                 __FILE__, __LINE__, hipGetErrorString( err ) );
        exit( -1 );
        // or 
        // return false;
     }

  kstatus = hipDeviceSynchronize();
  if (kstatus) {
    std::cout << "cuda_setup_kernel status = " << kstatus << "\n";
    return false;
  }

  return true;
}
#endif

