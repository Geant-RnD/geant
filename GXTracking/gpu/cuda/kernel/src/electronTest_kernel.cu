#include "hip/hip_runtime.h"
#include "electronTest_kernel.h"

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//Common
#include "GXTrack.h"
#include "GPPhysicsTable.h"
#include "GPPhysics2DVector.h"

//Material
#include "GPElement.h"
#include "GPMaterial.h"
#include "GPVParticleChange.h"

//EMPhysics
#include "GPeBremsstrahlung.h"
#include "GPSeltzerBergerRelModel.h"
#include "GPeIonisation.h"
#include "GPMollerBhabhaModel.h"
#include "GPIonisParamMat.h"
#include "GPUniversalFluctuation.h"
#include "GPeMultipleScattering.h"
#include "GPUrbanMscModel95.h"

#include "stdio.h"

FQUALIFIER
G4double GetKineticEnergy(GXTrack *track) 
{
  G4double p = sqrt(track->px*track->px + track->py*track->py 
		  + track->pz*track->pz);
  G4double mass = electron_mass_c2*track->q*track->q;
  G4double ekin = p*p/(sqrt(p*p + mass*mass) + mass);
  return ekin;
}

FQUALIFIER
void DefineMaterial(GPMaterial *aMat) 
{
  //PbWO4 aka CMS crystal
  GPMaterial_Constructor_ByElement(aMat,8.28*g/cm3);

  GPElement ele_Pb;
  GPElement ele_W;
  GPElement ele_O4;

  GPElement_Constructor(&ele_Pb,82,207.2*g/mole);
  GPElement_Constructor(&ele_W,74,183.84*g/mole);
  GPElement_Constructor(&ele_O4,8*4,15.9994*4*g/mole);

  GPMaterial_AddElement(aMat,ele_Pb,0.45532661);
  GPMaterial_AddElement(aMat,ele_W,0.40403397);
  GPMaterial_AddElement(aMat,ele_O4,0.14063942);
}

FQUALIFIER
void StackSecondaries(GPVParticleChange *particleChange,
                      GXTrack *secTracks,
                      G4int *stackSize,
                      G4int *offset) 
{
  //store secondaries in a global stack (fixed size memory)
  G4int nsec = particleChange->GetNumberOfSecondaries();

  for(int isec = 0 ; isec < nsec ; ++isec) {
    //offset is a global counter for the last array position of secondaries 
    GXTrack* aSecTrack = particleChange->GetSecondary(isec);
#ifndef __CUDA_ARCH__
    secTracks[*stackSize] = *aSecTrack;
    ++(*stackSize);
#else
    *offset = atomicAdd(stackSize,1);
    secTracks[*offset] = *aSecTrack;
#endif
  }
}

FQUALIFIER
void Print(GXTrack *track, GPVParticleChange *particleChange)
{
  G4int nsec = particleChange->GetNumberOfSecondaries();
#ifndef __CUDA_ARCH__
  printf("CPU (step,ekin,nsec) = (%f,%f,%d)\n",track->s,track->E,nsec);
#else
  printf("GPU (step,ekin,nsec) = (%f,%f,%d)\n",track->s,track->E,nsec);
#endif
}

//-----------------------------------------------------------------------------
// Standard Electron Processes
//-----------------------------------------------------------------------------

GLOBALFUNC
void electron_kernel(hiprandState* devStates,
		     GXTrack *track, size_t nTrackSize,
		     GPPhysicsTable* eBrem_table, 
		     GPPhysicsTable* eIoni_table, 
		     GPPhysicsTable* eIoni_range, 
		     GPPhysicsTable* eIoni_dedx, 
		     GPPhysicsTable* eIoni_invr, 
		     GPPhysicsTable* msc_table,
		     GPPhysics2DVector* sbData,
		     GXTrack *secTracks, G4int *stackSize, G4int *offset,
		     G4int isStack, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  GPForceCondition condition;
  GPGPILSelection  selection;

  G4double step  = 0;
  G4double step1  = 0;
  G4double step_brem = 0;
  G4double step_ioni = 0;
  G4double step_msc  = 0;
  G4double ekin  = 0;
  G4double proposedStep = 0;

  //EM processes and models
  GPSeltzerBergerRelModel aModel(devStates,tid,sbData);
  GPeBremsstrahlung brem(devStates,tid, eBrem_table);
  brem.InitialiseProcess(&aModel);

  GPMollerBhabhaModel bModel(devStates,tid);
  GPeIonisation ioni(devStates,tid, eIoni_table, eIoni_range,
                             eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&bModel);

  GPUrbanMscModel95 cModel(devStates,tid);
  GPeMultipleScattering msc(devStates,tid,msc_table);
  msc.InitialiseProcess(&cModel);

  GPVParticleChange aPartChange;

  while (tid < nTrackSize) {

    ekin = GetKineticEnergy(&track[tid]);
    step = track[tid].s;

    brem.StartTracking();
    step_brem = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
                                                          &condition);

    ioni.StartTracking();
    step_ioni = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
                                                          &condition);
    msc.StartTracking();
    step_msc = msc.PostStepGetPhysicalInteractionLength(&track[tid],&condition);

    //physics model defining the current step
    unsigned int model = 0;

    if(step_brem > step_ioni) model = 1;   
    if(step_ioni > step_msc) model = 2;  

    switch (model) {
    case 0 : 
      proposedStep = step_brem;
      step1 = brem.AlongStepGetPhysicalInteractionLength(&selection);
      if(step1 < proposedStep) proposedStep = step1;
      aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
      aPartChange = brem.PostStepDoIt(&track[tid],&aMat);
      break;
    case 1 :
      proposedStep = step_ioni;
      step1 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
      if(step1 < proposedStep) proposedStep = step1;
      aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
      aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);
      break;
    case 2 :
      proposedStep = step_msc;                         
      if(step1 < proposedStep) proposedStep = step1;
      step1 = msc.AlongStepGetPhysicalInteractionLength(&aMat,ekin,step,
                                                        &selection);
      aPartChange = msc.AlongStepDoIt(&aMat,&track[tid]);
      aPartChange = msc.PostStepDoIt(&track[tid]);

      break;
    default :
      break;
    }

    track[tid].s = proposedStep;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,offset);
    //    if(runType==1) Print(&track[tid], &aPartChange);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void electron_gpu(hiprandState* devStates,
		  GXTrack *track, size_t nTrackSize,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* eIoni_range, 
		  GPPhysicsTable* eIoni_dedx, 
		  GPPhysicsTable* eIoni_invr, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData,
		  GXTrack *secTracks, G4int *stackSize, G4int *offset,
		  G4int isStack, G4int runType, 
		  int blocksPerGrid, int threadsPerBlock) 
{
  electron_kernel<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
                  nTrackSize,eBrem_table,eIoni_table,
                  eIoni_range,eIoni_dedx,eIoni_invr, msc_table,
		  sbData,secTracks,stackSize,offset,isStack,runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void electron_cpu(GXTrack *track, size_t nTrackSize,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* eIoni_range, 
		  GPPhysicsTable* eIoni_dedx, 
		  GPPhysicsTable* eIoni_invr, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData,
		  GXTrack *secTracks, G4int *stackSize,
		  G4int isStack, G4int runType)
{
  GPMaterial aMat;
  DefineMaterial(&aMat);

  GPForceCondition condition;
  GPGPILSelection  selection;

  G4double step  = 0;
  G4double step1  = 0;
  G4double step_brem = 0;
  G4double step_ioni = 0;
  G4double step_msc  = 0;
  G4double ekin  = 0;
  G4double proposedStep = 0;

  //EM processes and models
  GPSeltzerBergerRelModel aModel(0,-1,sbData);
  GPeBremsstrahlung brem(0,-1, eBrem_table);
  brem.InitialiseProcess(&aModel);

  GPMollerBhabhaModel bModel(0,-1);
  GPeIonisation ioni(0,-1, eIoni_table, eIoni_range,
                             eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&bModel);

  GPUrbanMscModel95 cModel(0,-1);
  GPeMultipleScattering msc(0,-1,msc_table);
  msc.InitialiseProcess(&cModel);

  GPVParticleChange aPartChange;

  for (int tid = 0; tid < nTrackSize; tid++) {

    ekin = GetKineticEnergy(&track[tid]);
    step = track[tid].s;

    brem.StartTracking();
    step_brem = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
                                                          &condition);
    ioni.StartTracking();
    step_ioni = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
                                                          &condition);
    msc.StartTracking();
    step_msc = msc.PostStepGetPhysicalInteractionLength(&track[tid],&condition);

    //physics model defining the current step
    unsigned int model = 0;

    if(step_brem > step_ioni) model = 1;   
    if(step_ioni > step_msc) model = 2;  

    switch (model) {
    case 0 : 
      proposedStep = step_brem;
      step1 = brem.AlongStepGetPhysicalInteractionLength(&selection);
      if(step1 < proposedStep) proposedStep = step1;
      aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
      aPartChange = brem.PostStepDoIt(&track[tid],&aMat);
      break;
    case 1 :
      proposedStep = step_ioni;
      step1 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
      if(step1 < proposedStep) proposedStep = step1;
      aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
      aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);
      break;
    case 2 :
      proposedStep = step_msc;                         
      step1 = msc.AlongStepGetPhysicalInteractionLength(&aMat,ekin,step,
                                                        &selection);
      if(step1 < proposedStep) proposedStep = step1;
      aPartChange = msc.AlongStepDoIt(&aMat,&track[tid]);
      aPartChange = msc.PostStepDoIt(&track[tid]);
      break;
    default :
      break;
    }

    track[tid].s = proposedStep;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,0);
    //    if(runType==1) Print(&track[tid], &aPartChange);
  }
}


//-----------------------------------------------------------------------------
// GPeBremsstrahlungProcess + Fixed size memory for secondaies
//-----------------------------------------------------------------------------

GLOBALFUNC
void brem_kernel(hiprandState* devStates, 
		 GXTrack *track, size_t nTracks,
		 GPPhysicsTable* eBrem_table, 
		 GPPhysicsTable* eIoni_table, 
		 GPPhysicsTable* msc_table,
		 GPPhysics2DVector* sbData,
		 GXTrack *secTracks, G4int *stackSize, G4int *offset, 
		 G4int isStack, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPSeltzerBergerRelModel aModel(devStates,tid,sbData);
  GPeBremsstrahlung brem(devStates,tid,eBrem_table);
  brem.InitialiseProcess(&aModel);

  //local varialbles
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  while (tid < nTracks) {

    step = track[tid].s;
    brem.StartTracking();

    step1 = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = brem.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = brem.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,offset);
    //    if(runType==1) Print(&track[tid], &aPartChange);

    tid += blockDim.x * gridDim.x;
  }

  //ensure that all threads complete before copying
  __syncthreads();

}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void brem_gpu(hiprandState* devStates,
	      GXTrack *track, size_t nTrackSize,
	      GPPhysicsTable* eBrem_table, 
	      GPPhysicsTable* eIoni_table, 
	      GPPhysicsTable* msc_table,
	      GPPhysics2DVector* sbData,
	      GXTrack *secTrack, G4int *stackSize, G4int *offset, 
	      G4int isStack, G4int runType,
	      int blocksPerGrid, int threadsPerBlock) 
{
  brem_kernel<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
		       nTrackSize,eBrem_table,eIoni_table,msc_table,
		       sbData,secTrack,stackSize,offset,isStack,runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void brem_cpu(GXTrack *track, size_t nTracks,
	      GPPhysicsTable* eBrem_table, 
	      GPPhysicsTable* eIoni_table, 
	      GPPhysicsTable* msc_table,
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize,
	      G4int isStack, G4int runType)
{
  GPMaterial aMat;
  DefineMaterial(&aMat);


  //EM models and processes
  GPSeltzerBergerRelModel aModel(0,-1,sbData);
  GPeBremsstrahlung brem(0,-1, eBrem_table);
  brem.InitialiseProcess(&aModel);

  //local variables
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  for (int tid = 0; tid < nTracks; ++tid) {

    step = track[tid].s;
    brem.StartTracking();

    step1 = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = brem.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = brem.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,0);
    //    if(runType==1) Print(&track[tid], &aPartChange);

  }
}

//-----------------------------------------------------------------------------
// GPeBremsstrahlungProcess + Dynamic memory for secondaies
//-----------------------------------------------------------------------------

GLOBALFUNC
void brem_kernel_dma(hiprandState* devStates, GXTrack *track, 
		     size_t nTracks,
		     GPPhysicsTable* eBrem_table, 
		     GPPhysicsTable* eIoni_table, 
		     GPPhysicsTable* msc_table,
		     GPPhysics2DVector* sbData,
		     GXSecContainer *container,
		     G4int *stackSize,
		     G4int *offset) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPSeltzerBergerRelModel aModel(devStates,tid,sbData);
  GPeBremsstrahlung brem(devStates,tid,eBrem_table);
  brem.InitialiseProcess(&aModel);

  //local varialbles
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  //secondary
  __shared__ GXTrack* secBuffer;
  __shared__ int* tsec;

  G4int maxSec = (nTracks%gridDim.x !=0 ) ?
    (nTracks/gridDim.x + 1)*maxSecondaryPerStep : 
    (nTracks/gridDim.x)*maxSecondaryPerStep ;

  if(threadIdx.x ==0 ) {
    secBuffer = (GXTrack*)malloc(maxSec*sizeof(GXTrack));
    tsec = (int*)malloc(sizeof(int));
    *tsec = 0;
  }

  __syncthreads();

  while (tid < nTracks) {

    step = track[tid].s;
    brem.StartTracking();

    step1 = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = brem.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = brem.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    //secondary collection
    GXTrack aGamma = aModel.GetSecondary();
    secBuffer[atomicAdd(tsec,1)] = aGamma;

    tid += blockDim.x * gridDim.x;
  }

  //ensure that all threads complete before copying
  __syncthreads();

  //store the size/address of secondaries and copy them to the global memory
  if(threadIdx.x ==0 ) {

    G4int blockIndex = blockIdx.x;
    atomicAdd(stackSize,*tsec);

    container[blockIndex].size = *tsec;
    container[blockIndex].addr = (GXTrack*)malloc((*tsec)*sizeof(GXTrack));
    memcpy(container[blockIndex].addr,secBuffer,(*tsec)*sizeof(GXTrack));
 
    //free the shared memory
    free(secBuffer);
    free(tsec);
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void brem_gpu_dma(hiprandState* devStates, GXTrack *track, size_t nTrackSize,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData,
		  GXSecContainer *secContainer_d,
		  G4int *stackSize,
		  G4int *offset,
		  int blocksPerGrid, int threadsPerBlock) 
{
  brem_kernel_dma<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
		     nTrackSize,eBrem_table,eIoni_table,msc_table,
		     sbData,secContainer_d,stackSize,offset);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void brem_cpu_dma(GXTrack *track, size_t nTracks,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData)
{

  GPMaterial aMat;
  DefineMaterial(&aMat);

  GPForceCondition condition;
  GPGPILSelection  selection;

  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;

  //EM models and processes
  GPSeltzerBergerRelModel aModel(0,-1,sbData);
  GPeBremsstrahlung brem(0,-1, eBrem_table);
  brem.InitialiseProcess(&aModel);

  GPVParticleChange aPartChange;

  //secondary
  G4int stackSize = 0;

  GXSecContainer *container 
    = (GXSecContainer*) malloc(nTracks*sizeof(GXSecContainer));

  for (int tid = 0; tid < nTracks; ++tid) {

    step = track[tid].s;
    brem.StartTracking();

    step1 = brem.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = brem.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = brem.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = brem.PostStepDoIt(&track[tid],&aMat);

    //    printf("CPU-Brem (step1,step2) = (%f,%f)\n",step1,step2);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    //secondary collection
    G4int nsec = aPartChange.GetNumberOfSecondaries();
    GXTrack* secondaries = (GXTrack*)malloc(nsec*sizeof(GXTrack));

    for(int isec = 0 ; isec < nsec ; ++isec) {
      GXTrack* aSecTrack = aPartChange.GetSecondary(isec);
      secondaries[isec] = *aSecTrack;
    }

    container[tid].size = nsec;
    container[tid].addr = secondaries;
    stackSize += nsec;

  }

  GXTrack *secTracks = (GXTrack*)malloc(stackSize*sizeof(GXTrack));
  G4int offset = 0;

  for(int it = 0 ; it < nTracks ; ++it) {
    memcpy(secTracks+offset,container[it].addr,
           container[it].size*sizeof(GXTrack));
    free(container[it].addr);
    offset += container[it].size; 
  }

  free(secTracks);
}

//-----------------------------------------------------------------------------
// GPeIonisationProcess + Preallocated fixed memory for secondaries
//-----------------------------------------------------------------------------

GLOBALFUNC
void ioni_kernel(hiprandState* devStates, GXTrack *track, 
		 size_t nTracks,
		 GPPhysicsTable* eBrem_table, 
		 GPPhysicsTable* eIoni_table, 
		 GPPhysicsTable* eIoni_range, 
		 GPPhysicsTable* eIoni_dedx, 
		 GPPhysicsTable* eIoni_invr, 
		 GPPhysicsTable* msc_table,
		 GPPhysics2DVector* sbData,
		 GXTrack *secTracks, G4int *stackSize, G4int *offset, 
		 G4int isStack, G4int runType)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPMollerBhabhaModel aModel(devStates,tid);
  GPeIonisation ioni(devStates,tid, eIoni_table, eIoni_range,
		     eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&aModel);

  //local variables
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  while (tid < nTracks) {

    step = track[tid].s;
    ioni.StartTracking();

    step1 = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,offset);
    //    if(runType==1) Print(&track[tid], &aPartChange);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void ioni_gpu(hiprandState* devStates, GXTrack *track, size_t nTrackSize,
	      GPPhysicsTable* eBrem_table, 
	      GPPhysicsTable* eIoni_table, 
	      GPPhysicsTable* eIoni_range, 
	      GPPhysicsTable* eIoni_dedx, 
	      GPPhysicsTable* eIoni_invr, 
	      GPPhysicsTable* msc_table,
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize, G4int *offset, 
	      G4int isStack, G4int runType, 
	      int blocksPerGrid, int threadsPerBlock) 
{
  ioni_kernel<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
		       nTrackSize,eBrem_table,eIoni_table,
	               eIoni_range,eIoni_dedx,eIoni_invr,msc_table,
		       sbData,secTracks,stackSize,offset,isStack,runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void ioni_cpu(GXTrack *track, size_t nTracks,
	      GPPhysicsTable* eBrem_table, 
	      GPPhysicsTable* eIoni_table, 
	      GPPhysicsTable* eIoni_range, 
	      GPPhysicsTable* eIoni_dedx, 
	      GPPhysicsTable* eIoni_invr, 
	      GPPhysicsTable* msc_table,
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize,
	      G4int isStack, G4int runType)
{

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPMollerBhabhaModel aModel(0,-1);
  GPeIonisation ioni(0,-1, eIoni_table, eIoni_range,
		     eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&aModel);

  //local variables
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  for (int tid = 0; tid < nTracks; ++tid) {

    step = track[tid].s;
    ioni.StartTracking();

    step1 = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,0);
    //    if(runType==1) Print(&track[tid], &aPartChange);
  }
}

//-----------------------------------------------------------------------------
// GPeIonisationProcess + Dynamic memory for secondaries
//-----------------------------------------------------------------------------

GLOBALFUNC
void ioni_kernel_dma(hiprandState* devStates, GXTrack *track, 
		     size_t nTracks,
		     GPPhysicsTable* eBrem_table, 
		     GPPhysicsTable* eIoni_table, 
		     GPPhysicsTable* eIoni_range, 
		     GPPhysicsTable* eIoni_dedx, 
		     GPPhysicsTable* eIoni_invr, 
		     GPPhysicsTable* msc_table,
		     GPPhysics2DVector* sbData,
		     GXSecContainer *container,
		     G4int *stackSize,
		     G4int *offset) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPMollerBhabhaModel aModel(devStates,tid);
  GPeIonisation ioni(devStates,tid, eIoni_table, eIoni_range,
		     eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&aModel);

  //local variables
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  //secondary
  __shared__ GXTrack* secBuffer;
  __shared__ int* tsec;

  G4int maxSec = (nTracks%gridDim.x !=0 ) ?
    (nTracks/gridDim.x + 1)*maxSecondaryPerStep : 
    (nTracks/gridDim.x)*maxSecondaryPerStep ;

  if(threadIdx.x ==0 ) {
    secBuffer = (GXTrack*)malloc(maxSec*sizeof(GXTrack));
    tsec = (int*)malloc(sizeof(int));
    *tsec = 0;
  }

  __syncthreads();

  //  G4int nsec = 0;
  //  GXTrack aTrack;

  while (tid < nTracks) {

    step = track[tid].s;
    ioni.StartTracking();

    step1 = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    //secondary
    GXTrack aSecTrack = aModel.GetSecondary();
    secBuffer[atomicAdd(tsec,1)] = aSecTrack;

    tid += blockDim.x * gridDim.x;
  }

  //ensure that all threads complete before copying
  __syncthreads();

  //store the size/address of secondaries and copy them to the global memory
  if(threadIdx.x ==0 ) {

    G4int blockIndex = blockIdx.x;
    atomicAdd(stackSize,*tsec);

    container[blockIndex].size = *tsec;
    container[blockIndex].addr = (GXTrack*)malloc((*tsec)*sizeof(GXTrack));
    memcpy(container[blockIndex].addr,secBuffer,(*tsec)*sizeof(GXTrack));
 
    //free the shared memory
    free(secBuffer);
    free(tsec);
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void ioni_gpu_dma(hiprandState* devStates, GXTrack *track, size_t nTrackSize,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* eIoni_range, 
		  GPPhysicsTable* eIoni_dedx, 
		  GPPhysicsTable* eIoni_invr, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData,
		  GXSecContainer *secContainer_d,
		  G4int *stackSize,
		  G4int *offset,
		  int blocksPerGrid, int threadsPerBlock) 
{
  ioni_kernel_dma<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
		       nTrackSize,eBrem_table,eIoni_table,
	               eIoni_range,eIoni_dedx,eIoni_invr,
                       msc_table,
		       sbData,secContainer_d,stackSize,offset);
}

void ioni_cpu_dma(GXTrack *track, size_t nTracks,
		  GPPhysicsTable* eBrem_table, 
		  GPPhysicsTable* eIoni_table, 
		  GPPhysicsTable* eIoni_range, 
		  GPPhysicsTable* eIoni_dedx, 
		  GPPhysicsTable* eIoni_invr, 
		  GPPhysicsTable* msc_table,
		  GPPhysics2DVector* sbData)
{

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPMollerBhabhaModel aModel(0,-1);
  GPeIonisation ioni(0,-1, eIoni_table, eIoni_range,
		     eIoni_dedx, eIoni_invr);
  ioni.InitialiseProcess(&aModel);

  //local variables
  G4double step  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  //secondary
  G4int stackSize = 0;

  GXSecContainer *container 
    = (GXSecContainer*) malloc(nTracks*sizeof(GXSecContainer));

  for (int tid = 0; tid < nTracks; ++tid) {

    step = track[tid].s;
    ioni.StartTracking();

    step1 = ioni.PostStepGetPhysicalInteractionLength(&track[tid],step,
						      &condition);
    step2 = ioni.AlongStepGetPhysicalInteractionLength(&selection);
    aPartChange = ioni.AlongStepDoIt(&track[tid],&aMat,1*cm);
    aPartChange = ioni.PostStepDoIt(&track[tid],&aMat);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    //secondary collection
    G4int nsec = aPartChange.GetNumberOfSecondaries();;
    GXTrack* secondaries = (GXTrack*)malloc(nsec*sizeof(GXTrack));

    for(int isec = 0 ; isec < nsec ; ++isec) {
      GXTrack* aSecTrack = aPartChange.GetSecondary(isec);
      secondaries[isec] = *aSecTrack;
    }

    container[tid].size = nsec;
    container[tid].addr = secondaries;
    stackSize += nsec;

  }

  GXTrack *secTracks = (GXTrack*)malloc(stackSize*sizeof(GXTrack));
  G4int offset = 0;

  for(int it = 0 ; it < nTracks ; ++it) {
    memcpy(secTracks+offset,container[it].addr,
           container[it].size*sizeof(GXTrack));
    free(container[it].addr);
    offset += container[it].size; 
  }

  free(secTracks);
}

//-----------------------------------------------------------------------------
// GPeMultipleScatteringProcess
//-----------------------------------------------------------------------------

GLOBALFUNC
void msc_kernel(hiprandState* devStates, GXTrack *track, 
		size_t nTrackSize,
		GPPhysicsTable* eBrem_table, 
		GPPhysicsTable* eIoni_table, 
		GPPhysicsTable* msc_table,
		GPPhysics2DVector* sbData,
		GXTrack *secTracks, G4int *stackSize, G4int *offset, 
		G4int isStack, G4int runType)
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPUrbanMscModel95 cModel(devStates,tid);
  GPeMultipleScattering msc(devStates,tid,msc_table);
  msc.InitialiseProcess(&cModel);

  //local vairables
  G4double step  = 0;
  G4double ekin  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  while (tid < nTrackSize) {

    ekin = GetKineticEnergy(&track[tid]);
    step = track[tid].s;
    msc.StartTracking();

    step1 = msc.PostStepGetPhysicalInteractionLength(&track[tid],&condition);
    step2 = msc.AlongStepGetPhysicalInteractionLength(&aMat,ekin,step,
    						      &selection);
    aPartChange = msc.AlongStepDoIt(&aMat,&track[tid]);
    aPartChange = msc.PostStepDoIt(&track[tid]);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,offset);
    //    if(runType==1) Print(&track[tid], &aPartChange);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void msc_gpu(hiprandState* devStates,
	     GXTrack *track, size_t nTrackSize,
	     GPPhysicsTable* eBrem_table, 
	     GPPhysicsTable* eIoni_table, 
	     GPPhysicsTable* msc_table,
	     GPPhysics2DVector* sbData,
	     GXTrack *secTracks, G4int *stackSize, G4int *offset, 
	     G4int isStack, G4int runType,
	     int blocksPerGrid, int threadsPerBlock) 
{
  msc_kernel<<< blocksPerGrid, threadsPerBlock >>>(devStates,track,
		       nTrackSize,eBrem_table,eIoni_table,msc_table,
		       sbData,secTracks,stackSize,offset,isStack,runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void msc_cpu(GXTrack *track, size_t nTrackSize,
	     GPPhysicsTable* eBrem_table, 
	     GPPhysicsTable* eIoni_table, 
	     GPPhysicsTable* msc_table,
	     GPPhysics2DVector* sbData,
	     GXTrack *secTracks, G4int *stackSize,
	     G4int isStack, G4int runType)
{
  GPMaterial aMat;
  DefineMaterial(&aMat);

  //EM models and processes
  GPUrbanMscModel95 cModel(0,-1);
  GPeMultipleScattering msc(0,-1,msc_table);
  msc.InitialiseProcess(&cModel);

  //local vairables
  G4double step  = 0;
  G4double ekin  = 0;
  G4double step1 = 0;
  G4double step2 = 0;
  GPForceCondition condition;
  GPGPILSelection  selection;
  GPVParticleChange aPartChange;

  for (int tid = 0; tid < nTrackSize; ++tid) {

    ekin = GetKineticEnergy(&track[tid]);
    step = track[tid].s;
    msc.StartTracking();

    step1 = msc.PostStepGetPhysicalInteractionLength(&track[tid],&condition);
    step2 = msc.AlongStepGetPhysicalInteractionLength(&aMat,ekin,step,
    						      &selection);
    aPartChange = msc.AlongStepDoIt(&aMat,&track[tid]);
    aPartChange = msc.PostStepDoIt(&track[tid]);

    track[tid].s = (step1 < step2) ? step1 : step2;
    track[tid].E = aPartChange.GetProposedKineticEnergy();

    if(isStack==0) StackSecondaries(&aPartChange,secTracks,stackSize,0);
    //    if(runType==1) Print(&track[tid], &aPartChange);

  }
}

//-----------------------------------------------------------------------------
// Other kernels
//-----------------------------------------------------------------------------
#include "dma_kernel.cu"

//-----------------------------------------------------------------------------
// Common
//-----------------------------------------------------------------------------
#include "GPStep.cu"
#include "GPStepPoint.cu"
#include "GPThreeVector.cu"
#include "GPPhysicsTable.cu"
#include "GPVParticleChange.cu"

//-----------------------------------------------------------------------------
// GPMaterial
//-----------------------------------------------------------------------------
#include "GPElement.cu"
#include "GPMaterial.cu"
#include "GPIonisParamMat.cu"

//EMPhysics
#include "GPeBremsstrahlung.cu"
#include "GPSeltzerBergerRelModel.cu"
#include "GPPhysics2DVector.cu"
#include "GPeIonisation.cu"
#include "GPMollerBhabhaModel.cu"
#include "GPUniversalFluctuation.cu"
#include "GPeMultipleScattering.cu"
#include "GPUrbanMscModel95.cu"
