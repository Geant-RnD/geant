#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

//Common
#include "GXTrack.h"
#include "GPPhysicsTable.h"
#include "GPPhysicsTableType.h"
#include "GPPhysics2DVector.h"
#include "GPForceCondition.h"
#include "GPVParticleChange.h"

//Material
#include "GPElement.h"
#include "GPMaterial.h"
#include "GPSandiaTable.h"

//Electron Processes/Models
#include "GPeBremsstrahlung.h"
#include "GPSeltzerBergerRelModel.h"
#include "GPeIonisation.h"
#include "GPMollerBhabhaModel.h"
#include "GPIonisParamMat.h"
#include "GPUniversalFluctuation.h"
#include "GPeMultipleScattering.h"
#include "GPUrbanMscModel95.h"

//Photon Processes/Models
#include "GPPhotonModel.h"
#include "GPPhotonProcess.h"
#include "GPEmProcessType.h"

//Manager
#include "GPProcessManager.h"
#include "GPTrackingManager.h"
#include "GPSteppingManager.h"

//Geometry
#include "GPGeomManager.h"
#include "GPVPhysicalVolume.h"
#include "GPTouchableHistory.h"
#include "GPNavigator.h"

//Transportation
#include "GXFieldMap.h"
#include "GXFieldTrack.h"
#include "GXEquationOfMotion.h"
#include "GXClassicalRK4.h"
#include "GXMagIntegratorDriver.h"
#include "GXChordFinder.h"
#include "GXFieldManager.h"
#include "GXPropagatorInField.h"
#include "GXMultiLevelLocator.h"
#include "GXTransportation.h"

#include "stdio.h"

//-----------------------------------------------------------------------------
// transportation Kernel - no physics processes
//-----------------------------------------------------------------------------

GLOBALFUNC
void trans_kernel(hiprandState* devStates,
		  GXTrack *track, size_t nTrackSize,
		  GPGeomManager *geomManager,
		  GXFieldMap *magMap,
		  GPPhysicsTable* physicsTable, 
		  GPPhysics2DVector* sbData,
		  GXTrack *secTracks, G4int *stackSize, G4int *offset,
		  G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(devStates,tid,sbData);
  GPeBremsstrahlung bremProcess(devStates,tid,
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  GPMollerBhabhaModel ioniModel(devStates,tid);
  GPeIonisation ioniProcess(devStates,tid, &physicsTable[kLambda_eIoni], 
                            &physicsTable[kRange_eIoni], 
                            &physicsTable[kDEDX_eIoni], 
                            &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  GPUrbanMscModel95 mscModel(devStates,tid);
  GPeMultipleScattering mscProcess(devStates,tid,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(0,
					     0,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for msc_kernel
//-----------------------------------------------------------------------------
void trans_gpu(hiprandState* devStates,
	       GXTrack *track, size_t nTrackSize,
	       GPGeomManager *geomManager,
	       GXFieldMap *magMap,
	       GPPhysicsTable* physicsTable, 
	       GPPhysics2DVector* sbData,
	       GXTrack *secTracks, G4int *stackSize, G4int *offset,
	       G4int numStep, G4int runType, 
	       int blocksPerGrid, int threadsPerBlock,
	       hipStream_t stream) 
{
  trans_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize,  geomManager, magMap, physicsTable,
     sbData, secTracks, stackSize, offset, numStep,runType);
}
//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void trans_cpu(GXTrack *track, size_t nTrackSize,
	       GPGeomManager *geomManager,
	       GXFieldMap *magMap,
	       GPPhysicsTable* physicsTable, 
	       GPPhysics2DVector* sbData,
	       GXTrack *secTracks, G4int *stackSize,
	       G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(0,-1,sbData);
  GPeBremsstrahlung bremProcess(0,-1, 
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  GPMollerBhabhaModel ioniModel(0,-1);
  GPeIonisation ioniProcess(0,-1, &physicsTable[kLambda_eIoni], 
			    &physicsTable[kRange_eIoni], 
			    &physicsTable[kDEDX_eIoni], 
			    &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  GPUrbanMscModel95 mscModel(0,-1);
  GPeMultipleScattering mscProcess(0,-1,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(0,
					     0,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; tid++) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}

//-----------------------------------------------------------------------------
// brem Kernel - GPeBremsstrahlung/GPSeltzerBergerRelModel
//-----------------------------------------------------------------------------

GLOBALFUNC
void brem_kernel(hiprandState* devStates,
		 GXTrack *track, size_t nTrackSize,
		 GPGeomManager *geomManager,
		 GXFieldMap *magMap,
		 GPPhysicsTable* physicsTable, 
		 GPPhysics2DVector* sbData,
		 GXTrack *secTracks, G4int *stackSize, G4int *offset,
		 G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models

  GPSeltzerBergerRelModel bremModel(devStates,tid,sbData);
  GPeBremsstrahlung bremProcess(devStates,tid,
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  /*
  GPMollerBhabhaModel ioniModel(devStates,tid);
  GPeIonisation ioniProcess(devStates,tid, &physicsTable[kLambda_eIoni], 
                            &physicsTable[kRange_eIoni], 
                            &physicsTable[kDEDX_eIoni], 
                            &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  GPUrbanMscModel95 mscModel(devStates,tid);
  GPeMultipleScattering mscProcess(devStates,tid,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(&bremProcess,
					     0,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for brem_kernel
//-----------------------------------------------------------------------------
void brem_gpu(hiprandState* devStates,
	      GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize, G4int *offset,
	      G4int numStep, G4int runType, 
	      int blocksPerGrid, int threadsPerBlock,
	      hipStream_t stream) 
{
  brem_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize,  geomManager, magMap, physicsTable,
     sbData, secTracks, stackSize, offset, numStep,runType);
}
//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void brem_cpu(GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize,
	      G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  GPSeltzerBergerRelModel bremModel(0,-1,sbData);
  GPeBremsstrahlung bremProcess(0,-1, 
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  /*
  GPMollerBhabhaModel ioniModel(0,-1);
  GPeIonisation ioniProcess(0,-1, &physicsTable[kLambda_eIoni], 
			    &physicsTable[kRange_eIoni], 
			    &physicsTable[kDEDX_eIoni], 
			    &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  GPUrbanMscModel95 mscModel(0,-1);
  GPeMultipleScattering mscProcess(0,-1,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(&bremProcess,
					     0,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; tid++) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}
//-----------------------------------------------------------------------------
// ioni Kernel - GPeIonisation/GPMollerBhabhaModel
//-----------------------------------------------------------------------------

GLOBALFUNC
void ioni_kernel(hiprandState* devStates,
		 GXTrack *track, size_t nTrackSize,
		 GPGeomManager *geomManager,
		 GXFieldMap *magMap,
		 GPPhysicsTable* physicsTable, 
		 GPPhysics2DVector* sbData,
		 GXTrack *secTracks, G4int *stackSize, G4int *offset,
		 G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(devStates,tid,sbData);
  GPeBremsstrahlung bremProcess(devStates,tid,
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);
  */

  GPMollerBhabhaModel ioniModel(devStates,tid);
  GPeIonisation ioniProcess(devStates,tid, &physicsTable[kLambda_eIoni], 
                            &physicsTable[kRange_eIoni], 
                            &physicsTable[kDEDX_eIoni], 
                            &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  /*
  GPUrbanMscModel95 mscModel(devStates,tid);
  GPeMultipleScattering mscProcess(devStates,tid,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(0,
					     &ioniProcess,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for ioni_kernel
//-----------------------------------------------------------------------------
void ioni_gpu(hiprandState* devStates,
	      GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize, G4int *offset,
	      G4int numStep, G4int runType, 
	      int blocksPerGrid, int threadsPerBlock,
	      hipStream_t stream) 
{
  ioni_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize,  geomManager, magMap, physicsTable,
     sbData, secTracks, stackSize, offset, numStep,runType);
}
//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void ioni_cpu(GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GPPhysics2DVector* sbData,
	      GXTrack *secTracks, G4int *stackSize,
	      G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(0,-1,sbData);
  GPeBremsstrahlung bremProcess(0,-1, 
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);
  */

  GPMollerBhabhaModel ioniModel(0,-1);
  GPeIonisation ioniProcess(0,-1, &physicsTable[kLambda_eIoni], 
			    &physicsTable[kRange_eIoni], 
			    &physicsTable[kDEDX_eIoni], 
			    &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);

  /*
  GPUrbanMscModel95 mscModel(0,-1);
  GPeMultipleScattering mscProcess(0,-1,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);
  */

  aElectronProcessManager.AddElectronProcess(0,
					     &ioniProcess,
					     0);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; tid++) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}

//-----------------------------------------------------------------------------
// msc Kernel - GPeMultipleScattering/GPUrbanMscModel95
//-----------------------------------------------------------------------------

GLOBALFUNC
void msc_kernel(hiprandState* devStates,
		GXTrack *track, size_t nTrackSize,
		GPGeomManager *geomManager,
		GXFieldMap *magMap,
		GPPhysicsTable* physicsTable, 
		GPPhysics2DVector* sbData,
		GXTrack *secTracks, G4int *stackSize, G4int *offset,
		G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(devStates,tid,sbData);
  GPeBremsstrahlung bremProcess(devStates,tid,
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  GPMollerBhabhaModel ioniModel(devStates,tid);
  GPeIonisation ioniProcess(devStates,tid, &physicsTable[kLambda_eIoni], 
                            &physicsTable[kRange_eIoni], 
                            &physicsTable[kDEDX_eIoni], 
                            &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);
  */

  GPUrbanMscModel95 mscModel(devStates,tid);
  GPeMultipleScattering mscProcess(devStates,tid,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);

  aElectronProcessManager.AddElectronProcess(0,
					     0,
					     &mscProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);

    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for msc_kernel
//-----------------------------------------------------------------------------
void msc_gpu(hiprandState* devStates,
	     GXTrack *track, size_t nTrackSize,
	     GPGeomManager *geomManager,
	     GXFieldMap *magMap,
	     GPPhysicsTable* physicsTable, 
	     GPPhysics2DVector* sbData,
	     GXTrack *secTracks, G4int *stackSize, G4int *offset,
	     G4int numStep, G4int runType, 
	     int blocksPerGrid, int threadsPerBlock,
	     hipStream_t stream) 
{
  msc_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize,  geomManager, magMap, physicsTable,
     sbData, secTracks, stackSize, offset, numStep,runType);
}
//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void msc_cpu(GXTrack *track, size_t nTrackSize,
	     GPGeomManager *geomManager,
	     GXFieldMap *magMap,
	     GPPhysicsTable* physicsTable, 
	     GPPhysics2DVector* sbData,
	     GXTrack *secTracks, G4int *stackSize,
	     G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aElectronProcessManager;

  //EM processes and models
  /*
  GPSeltzerBergerRelModel bremModel(0,-1,sbData);
  GPeBremsstrahlung bremProcess(0,-1, 
				&physicsTable[kLambda_eBrem]);
  bremProcess.InitialiseProcess(&bremModel);

  GPMollerBhabhaModel ioniModel(0,-1);
  GPeIonisation ioniProcess(0,-1, &physicsTable[kLambda_eIoni], 
			    &physicsTable[kRange_eIoni], 
			    &physicsTable[kDEDX_eIoni], 
			    &physicsTable[kInverseRange_eIoni]);
  ioniProcess.InitialiseProcess(&ioniModel);
  */

  GPUrbanMscModel95 mscModel(0,-1);
  GPeMultipleScattering mscProcess(0,-1,&physicsTable[kLambda_msc]);
  mscProcess.InitialiseProcess(&mscModel);

  aElectronProcessManager.AddElectronProcess(0,
					     0,
					     &mscProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);
  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aElectronProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; tid++) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    aSteppingManager.SetTransportation(&transp,&aNavigator);
    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}

//-----------------------------------------------------------------------------
// compt Kernel 
//-----------------------------------------------------------------------------

GLOBALFUNC
void compt_kernel(hiprandState* devStates,
		  GXTrack *track, size_t nTrackSize,
		  GPGeomManager *geomManager,
		  GXFieldMap *magMap,
		  GPPhysicsTable* physicsTable, 
		  GXTrack *secTracks, G4int *stackSize, G4int *offset,
		  G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aPhotonProcessManager;

  //Construct photon processes/models
  GPPhotonModel comptModel(devStates,tid);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(devStates,tid,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);

  /*
  GPPhotonModel convModel(devStates,tid);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(devStates,tid,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);

  GPPhotonModel photModel(devStates,tid);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(devStates,tid,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);
  */

  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void compt_gpu(hiprandState* devStates,
	       GXTrack *track, size_t nTrackSize,
	       GPGeomManager *geomManager,
	       GXFieldMap *magMap,
	       GPPhysicsTable* physicsTable, 
	       GXTrack *secTracks, G4int *stackSize, G4int *offset,
	       G4int numStep, G4int runType,
	       int blocksPerGrid, int threadsPerBlock,
	       hipStream_t stream)
{
  compt_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize, geomManager, magMap, physicsTable,
     secTracks, stackSize, offset, numStep, runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void compt_cpu(GXTrack *track, size_t nTrackSize,
	       GPGeomManager *geomManager,
	       GXFieldMap *magMap,
	       GPPhysicsTable* physicsTable, 
	       GXTrack *secTracks, G4int *stackSize,
	       G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aPhotonProcessManager;

  GPPhotonModel comptModel(0,-1);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(0,-1,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);

  /*
  GPPhotonModel convModel(0,-1);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(0,-1,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);

  //PhotoElectricEffect processes
  GPPhotonModel photModel(0,-1);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(0,-1,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);
  */

  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; ++tid) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}

//-----------------------------------------------------------------------------
// conv Kernel 
//-----------------------------------------------------------------------------

GLOBALFUNC
void conv_kernel(hiprandState* devStates,
		 GXTrack *track, size_t nTrackSize,
		 GPGeomManager *geomManager,
		 GXFieldMap *magMap,
		 GPPhysicsTable* physicsTable, 
		 GXTrack *secTracks, G4int *stackSize, G4int *offset,
		 G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aPhotonProcessManager;

  //Construct photon processes/models
  /*
  GPPhotonModel comptModel(devStates,tid);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(devStates,tid,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);
  */

  GPPhotonModel convModel(devStates,tid);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(devStates,tid,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);

  /*
  GPPhotonModel photModel(devStates,tid);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(devStates,tid,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);
  */

  //  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void conv_gpu(hiprandState* devStates,
	      GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GXTrack *secTracks, G4int *stackSize, G4int *offset,
	      G4int numStep, G4int runType,
	      int blocksPerGrid, int threadsPerBlock,
	      hipStream_t stream)
{
  conv_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize, geomManager, magMap, physicsTable,
     secTracks, stackSize, offset, numStep, runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void conv_cpu(GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GXTrack *secTracks, G4int *stackSize,
	      G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aPhotonProcessManager;

  /*
  GPPhotonModel comptModel(0,-1);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(0,-1,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);
  */

  GPPhotonModel convModel(0,-1);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(0,-1,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);

  //PhotoElectricEffect processes
  /*
  GPPhotonModel photModel(0,-1);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(0,-1,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);
  */

  //  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; ++tid) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}

//-----------------------------------------------------------------------------
// compt Kernel 
//-----------------------------------------------------------------------------

GLOBALFUNC
void phot_kernel(hiprandState* devStates,
		 GXTrack *track, size_t nTrackSize,
		 GPGeomManager *geomManager,
		 GXFieldMap *magMap,
		 GPPhysicsTable* physicsTable, 
		 GXTrack *secTracks, G4int *stackSize, G4int *offset,
		 G4int numStep, G4int runType) 
{
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

  //process manager
  GPProcessManager aPhotonProcessManager;

  //Construct photon processes/models
  /*
  GPPhotonModel comptModel(devStates,tid);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(devStates,tid,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);

  GPPhotonModel convModel(devStates,tid);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(devStates,tid,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);
  */

  GPPhotonModel photModel(devStates,tid);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(devStates,tid,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);

  //  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  while (tid < nTrackSize) {

    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,offset);
    aTrackingManager.ProcessOneTrack(&track[tid]);
    tid += blockDim.x * gridDim.x;
  }
}

//-----------------------------------------------------------------------------
//  cuda wrapper for kernel
//-----------------------------------------------------------------------------
void phot_gpu(hiprandState* devStates,
	      GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GXTrack *secTracks, G4int *stackSize, G4int *offset,
	      G4int numStep, G4int runType,
	      int blocksPerGrid, int threadsPerBlock,
	      hipStream_t stream)
{
  phot_kernel<<< blocksPerGrid, threadsPerBlock, 0, stream >>>
    (devStates,track, nTrackSize, geomManager, magMap, physicsTable,
     secTracks, stackSize, offset, numStep, runType);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for CPU
//-----------------------------------------------------------------------------

void phot_cpu(GXTrack *track, size_t nTrackSize,
	      GPGeomManager *geomManager,
	      GXFieldMap *magMap,
	      GPPhysicsTable* physicsTable, 
	      GXTrack *secTracks, G4int *stackSize,
	      G4int numStep, G4int runType)
{
  //process manager
  GPProcessManager aPhotonProcessManager;

  /*
  GPPhotonModel comptModel(0,-1);
  comptModel.SetProcessType(kCompton);

  GPPhotonProcess comptProcess(0,-1,&physicsTable[kLambda_compt]);
  comptProcess.InitialiseProcess(kCompton,&comptModel);

  GPPhotonModel convModel(0,-1);
  convModel.SetProcessType(kConversion);

  GPPhotonProcess convProcess(0,-1,&physicsTable[kLambda_conv]);
  convProcess.InitialiseProcess(kConversion,&convModel);
  */

  //PhotoElectricEffect processes
  GPPhotonModel photModel(0,-1);
  photModel.SetProcessType(kPhotoElectric);

  GPSandiaTable aSandiaTable;
  GPSandiaTable_Constructor(&aSandiaTable);
  photModel.SetaSandiaTable(&aSandiaTable);

  GPPhotonProcess photProcess(0,-1,&physicsTable[kLambdaPrim_phot]);
  photProcess.InitialiseProcess(kPhotoElectric,&photModel);

  //  aPhotonProcessManager.AddPhotonProcess(comptProcess);  
  //  aPhotonProcessManager.AddPhotonProcess(convProcess);  
  aPhotonProcessManager.AddPhotonProcess(photProcess);  

  GPSteppingManager aSteppingManager(geomManager,magMap);

  GPStep aStep;
  aSteppingManager.SetStep(&aStep);

  GPTrackingManager aTrackingManager(&aPhotonProcessManager,
				     &aSteppingManager);
  aTrackingManager.SetNumberOfSteps(numStep);

  //Transportation
  GPVPhysicalVolume *world = geomManager->fWorldPhysical;

  GXMagneticField magField;
  GXMagneticField_Constructor(&magField,magMap);
  
  GXEquationOfMotion equaOfMotion;
  GXEquationOfMotion_Constructor(&equaOfMotion,&magField,-1.0);
  
  GXClassicalRK4 rk4;
  GXClassicalRK4_Constructor(&rk4,&equaOfMotion);
  
  GXMagInt_Driver magDriver;
  GXMagInt_Driver_Constructor(&magDriver,1.0,&rk4);

  GXChordFinder chordFinder;
  GXChordFinder_Constructor(&chordFinder,&magDriver);

  GXFieldManager aFieldManager;
  GXFieldManager_Constructor(&aFieldManager,&magField,&chordFinder, true);

  //Navigator
  GPNavigator aNavigator;
  GPNavigator_Constructor(&aNavigator);
  GPNavigator_SetWorldVolume(&aNavigator,world);

  //GPMultiLevelLocator
  GXMultiLevelLocator mLocator;

  //Propagator
  GXPropagatorInField propagatorInField;

  //Transporation
  GXTransportation transp;

  for (int tid = 0; tid < nTrackSize; ++tid) {
    //Geometry - Initialize Navigator and construct related structures
    GPNavigator_LocateGlobalPointAndSetup(&aNavigator,
					  GPThreeVector_create(track[tid].x,
							       track[tid].y,
							       track[tid].z),
                                          NULL,false,true);

    GXMultiLevelLocator_Constructor(&mLocator, &aNavigator);
    
    GXPropagatorInField_Constructor(&propagatorInField,&aNavigator,
                                    &aFieldManager,&mLocator); 
    
    GXTransportation_Constructor(&transp,&propagatorInField,0);

    //    aSteppingManager.SetTransportationProcess(&transp);
    aSteppingManager.SetTransportation(&transp,&aNavigator);

    aSteppingManager.SetSecondaryStack(secTracks,stackSize,0);
    aTrackingManager.ProcessOneTrack(&track[tid]);
  }
}


//-----------------------------------------------------------------------------
// Other kernels
//-----------------------------------------------------------------------------
#include "random_kernel.cu"

//-----------------------------------------------------------------------------
// Common
//-----------------------------------------------------------------------------
#include "GPThreeVector.cu"
#include "GPThreeVectorList.cu"
#include "GPRotationMatrix.cu"
#include "GPUtils.cu"
#include "GPPhysicsTable.cu"
#include "GPPhysics2DVector.cu"
#include "GPVParticleChange.cu"

//-----------------------------------------------------------------------------
// Material
//-----------------------------------------------------------------------------
#include "GPElement.cu"
#include "GPMaterial.cu"
#include "GPStep.cu"
#include "GPStepPoint.cu"
#include "GPAtomicShells.cu"
#include "GPSandiaTable.cu"
#include "GPIonisParamMat.cu"

//-----------------------------------------------------------------------------
// Electron processes/models
//-----------------------------------------------------------------------------
#include "GPeBremsstrahlung.cu"
#include "GPSeltzerBergerRelModel.cu"
#include "GPeIonisation.cu"
#include "GPMollerBhabhaModel.cu"
#include "GPUniversalFluctuation.cu"
#include "GPeMultipleScattering.cu"
#include "GPUrbanMscModel95.cu"

//-----------------------------------------------------------------------------
// Photon processes/models
//-----------------------------------------------------------------------------
#include "GPPhotonModel.cu"
#include "GPPhotonProcess.cu"

//-----------------------------------------------------------------------------
// Manager
//-----------------------------------------------------------------------------
#include "GPProcessManager.cu"
#include "GPTrackingManager.cu"
#include "GPSteppingManager.cu"

//-----------------------------------------------------------------------------
// GPGeometry
//-----------------------------------------------------------------------------
#include "GPAffineTransform.cu"
#include "GPAuxiliaryNavServices.cu"
#include "GPCombinedNavigation.cu"
#include "GPLineSection.cu"
#include "GPLogicalVolume.cu"
#include "GPNavigationLevel.cu"
#include "GPNavigator.cu"
#include "GPNavigationHistory.cu"
#include "GPNormalNavigation.cu"
#include "GPSmartVoxelHeader.cu"
#include "GPSmartVoxelNode.cu"
#include "GPSmartVoxelProxy.cu"
#include "GPTouchableHistory.cu"
#include "GPVoxelLimits.cu"
#include "GPVoxelNavigation.cu"
#include "GPVPhysicalVolume.cu"
#include "GPBox.cu"
#include "GPCons.cu"
#include "GPOrb.cu"
#include "GPTrd.cu"
#include "GPTubs.cu"
#include "GPVSolid.cu"
#include "GPChargeState.cu"
#include "GPUserGeometry.cu"
#include "GPVoxelHeader.cu"

//-----------------------------------------------------------------------------
// Magnetic Field and Transporation
//-----------------------------------------------------------------------------
#include "GXFieldTrack.cu"
#include "GXFieldManager.cu"
#include "GXMagneticField.cu"
#include "GXClassicalRK4.cu"
#include "GXEquationOfMotion.cu"
#include "GXMagIntegratorDriver.cu"
#include "GXChordFinder.cu"
#include "GXPropagatorInField.cu"
#include "GXMultiLevelLocator.cu"
#include "GXTransportation.cu"
