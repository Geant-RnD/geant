#include "hip/hip_runtime.h"
//
// ********************************************************************
// * License and Disclaimer                                           *
// *                                                                  *
// * The  Geant4 software  is  copyright of the Copyright Holders  of *
// * the Geant4 Collaboration.  It is provided  under  the terms  and *
// * conditions of the Geant4 Software License,  included in the file *
// * LICENSE and available at  http://cern.ch/geant4/license .  These *
// * include a list of copyright holders.                             *
// *                                                                  *
// * Neither the authors of this software system, nor their employing *
// * institutes,nor the agencies providing financial support for this *
// * work  make  any representation or  warranty, express or implied, *
// * regarding  this  software system or assume any liability for its *
// * use.  Please see the license in the file  LICENSE  and URL above *
// * for the full disclaimer and the limitation of liability.         *
// *                                                                  *
// * This  code  implementation is the result of  the  scientific and *
// * technical work of the GEANT4 collaboration.                      *
// * By using,  copying,  modifying or  distributing the software (or *
// * any work based  on the software)  you  agree  to acknowledge its *
// * use  in  resulting  scientific  publications,  and indicate your *
// * acceptance of all terms of the Geant4 Software license.          *
// ********************************************************************
//
//
// $Id: GPNystromRK4.cc,v 1.9 2010-09-10 15:42:09 japost Exp $
// GEANT4 tag $Name: not supported by cvs2svn $
//
// History:
// - Created:      I.Gavrilenko    15.05.2009   (as G4AtlasRK4)
// - Adaptations:  J. Apostolakis  May-Nov 2009
// -------------------------------------------------------------------

#include "GPNystromRK4.h"

//////////////////////////////////////////////////////////////////
// Constructor - with optional distance ( has default value)
//////////////////////////////////////////////////////////////////

FQUALIFIER
void GPNystromRK4_Constructor(GPNystromRK4 *This, 
			      GPEquationOfMotion* EqRhs, 
			      G4double distanceConstField)
{

  //  : G4MagIntegratorStepper(magEqRhs, 6),            // number of variables

  //EqRhs should be the one after GPEquationOfMotion_SetChargeMomentumMass is called

  GPNystromRK4_G4MagIntegratorStepper_Constructor(This,EqRhs,6,0);

  This->m_magdistance = distanceConstField;
  This->m_cof = 0.0;
  This->m_mom = 0.0;
  This->m_imom = 0.0;
  This->m_cachedMom = false; 

  This->m_fldPosition[0]  = This->m_iPoint[0] = This->m_fPoint[0] = This->m_mPoint[0] = 9.9999999e+99 ;
  This->m_fldPosition[1]  = This->m_iPoint[1] = This->m_fPoint[1] = This->m_mPoint[1] = 9.9999999e+99 ;
  This->m_fldPosition[2]  = This->m_iPoint[2] = This->m_fPoint[2] = This->m_mPoint[2] = 9.9999999e+99 ;
  This->m_fldPosition[3]  = -9.9999999e+99;
  This->m_lastField[0] = This->m_lastField[1] = This->m_lastField[2] = 0.0;

  This->m_magdistance2 = distanceConstField*distanceConstField;
}

////////////////////////////////////////////////////////////////
// Destructor
////////////////////////////////////////////////////////////////
/*
GPNystromRK4_Destructor()
{
}
*/
/////////////////////////////////////////////////////////////////////////////////
// Integration in one  step 
/////////////////////////////////////////////////////////////////////////////////

FQUALIFIER
void GPNystromRK4_Stepper(GPNystromRK4 *This,
			  const G4double P[],const G4double dPdS[],
			  G4double Step,G4double Po[],G4double Err[])
{
  G4double R[3] = {   P[0],   P[1] ,    P[2]};
  G4double A[3] = {dPdS[0], dPdS[1], dPdS[2]};

  This->m_iPoint[0]=R[0]; This->m_iPoint[1]=R[1]; This->m_iPoint[2]=R[2];

  const G4double one_sixth= 1./6.;
  G4double S  =     Step   ;
  G4double S5 =  .5*Step   ;
  G4double S4 = .25*Step   ;
  G4double S6 =     Step * one_sixth;   // Step / 6.;


  // John A  added, in order to emulate effect of call to changed/derived RHS
  // This->m_mom   = sqrt(P[3]*P[3]+P[4]*P[4]+P[5]*P[5]); 
  // This->m_imom  = 1./This->m_mom;
  // This->m_cof   = m_fEq->FCof()*This->m_imom;

  // Point 1
  //
  G4double K1[3] = { This->m_imom*dPdS[3], This->m_imom*dPdS[4], This->m_imom*dPdS[5] };
  
  // Point2
  //
  G4double p[4] = {R[0]+S5*(A[0]+S4*K1[0]),
		   R[1]+S5*(A[1]+S4*K1[1]),
		   R[2]+S5*(A[2]+S4*K1[2]),
		   P[7]                   }; 

  GPNystromRK4_getField(This, p);

  G4double A2[3] = {A[0]+S5*K1[0],A[1]+S5*K1[1],A[2]+S5*K1[2]};
  G4double K2[3] = {(A2[1]*This->m_lastField[2]-A2[2]*This->m_lastField[1])*This->m_cof,
		    (A2[2]*This->m_lastField[0]-A2[0]*This->m_lastField[2])*This->m_cof,
		    (A2[0]*This->m_lastField[1]-A2[1]*This->m_lastField[0])*This->m_cof};
 
  This->m_mPoint[0]=p[0]; This->m_mPoint[1]=p[1]; This->m_mPoint[2]=p[2];

  // Point 3 with the same magnetic field
  //
  G4double A3[3] = {A[0]+S5*K2[0],A[1]+S5*K2[1],A[2]+S5*K2[2]};
  G4double K3[3] = {(A3[1]*This->m_lastField[2]-A3[2]*This->m_lastField[1])*This->m_cof,
		    (A3[2]*This->m_lastField[0]-A3[0]*This->m_lastField[2])*This->m_cof,
		    (A3[0]*This->m_lastField[1]-A3[1]*This->m_lastField[0])*This->m_cof};
  
  // Point 4
  //
  p[0] = R[0]+S*(A[0]+S5*K3[0]);
  p[1] = R[1]+S*(A[1]+S5*K3[1]);
  p[2] = R[2]+S*(A[2]+S5*K3[2]);             

  GPNystromRK4_getField(This,p);
  
  G4double A4[3] = {A[0]+S*K3[0],A[1]+S*K3[1],A[2]+S*K3[2]};
  G4double K4[3] = {(A4[1]*This->m_lastField[2]-A4[2]*This->m_lastField[1])*This->m_cof,
		    (A4[2]*This->m_lastField[0]-A4[0]*This->m_lastField[2])*This->m_cof,
		    (A4[0]*This->m_lastField[1]-A4[1]*This->m_lastField[0])*This->m_cof};
  
  // New position
  //
  Po[0] = P[0]+S*(A[0]+S6*(K1[0]+K2[0]+K3[0]));
  Po[1] = P[1]+S*(A[1]+S6*(K1[1]+K2[1]+K3[1]));
  Po[2] = P[2]+S*(A[2]+S6*(K1[2]+K2[2]+K3[2]));

  This->m_fPoint[0]=Po[0]; This->m_fPoint[1]=Po[1]; This->m_fPoint[2]=Po[2];

  // New direction
  //
  Po[3] = A[0]+S6*(K1[0]+K4[0]+2.*(K2[0]+K3[0]));
  Po[4] = A[1]+S6*(K1[1]+K4[1]+2.*(K2[1]+K3[1]));
  Po[5] = A[2]+S6*(K1[2]+K4[2]+2.*(K2[2]+K3[2]));

  // Errors
  //
  Err[3] = S*fabs(K1[0]-K2[0]-K3[0]+K4[0]);
  Err[4] = S*fabs(K1[1]-K2[1]-K3[1]+K4[1]);
  Err[5] = S*fabs(K1[2]-K2[2]-K3[2]+K4[2]);
  Err[0] = S*Err[3]                       ;
  Err[1] = S*Err[4]                       ;
  Err[2] = S*Err[5]                       ;
  Err[3]*= This->m_mom                          ;
  Err[4]*= This->m_mom                          ;
  Err[5]*= This->m_mom                          ;

  // Normalize momentum
  //
  G4double normF = This->m_mom/sqrt(Po[3]*Po[3]+Po[4]*Po[4]+Po[5]*Po[5]);
  Po [3]*=normF; Po[4]*=normF; Po[5]*=normF; 

  // Pass Energy, time unchanged -- time is not integrated !!
  Po[6]=P[6]; Po[7]=P[7];
}


/////////////////////////////////////////////////////////////////////////////////
// Estimate the maximum distance from the curve to the chord
/////////////////////////////////////////////////////////////////////////////////

FQUALIFIER
G4double GPNystromRK4_DistChord(GPNystromRK4 *This) 
{
  G4double ax = This->m_fPoint[0]-This->m_iPoint[0];  
  G4double ay = This->m_fPoint[1]-This->m_iPoint[1];  
  G4double az = This->m_fPoint[2]-This->m_iPoint[2];
  G4double dx = This->m_mPoint[0]-This->m_iPoint[0]; 
  G4double dy = This->m_mPoint[1]-This->m_iPoint[1]; 
  G4double dz = This->m_mPoint[2]-This->m_iPoint[2];
  G4double d2 = (ax*ax+ay*ay+az*az)    ; 

  if(d2!=0.) {
    G4double  s = (ax*dx+ay*dy+az*dz)/d2;
    dx         -= (s*ax)                ;
    dy         -= (s*ay)                ;
    dz         -= (s*az)                ;
  }
  return sqrt(dx*dx+dy*dy+dz*dz);
}

/////////////////////////////////////////////////////////////////////////////////
// Derivatives calculation - caching the momentum value
/////////////////////////////////////////////////////////////////////////////////

FQUALIFIER
void GPNystromRK4_ComputeRightHandSide(GPNystromRK4 *This, 
				       const G4double P[], G4double dPdS[])
{
  G4double P4vec[4]= { P[0], P[1], P[2], P[7] }; // Time is P[7]
  GPNystromRK4_getField(This,P4vec);
  This->m_mom   = sqrt(P[3]*P[3]+P[4]*P[4]+P[5]*P[5])     ; 
  This->m_imom  = 1./This->m_mom                                ;
  This->m_cof   = GPEquationOfMotion_FCof(This->m_fEq)*This->m_imom                    ;
  This->m_cachedMom = true                                ; // Caching the value
  dPdS[0] = P[3]*This->m_imom                             ; // dx /ds
  dPdS[1] = P[4]*This->m_imom                             ; // dy /ds
  dPdS[2] = P[5]*This->m_imom                             ; // dz /ds
  dPdS[3] = This->m_cof*(P[4]*This->m_lastField[2]-P[5]*This->m_lastField[1]) ; // dPx/ds
  dPdS[4] = This->m_cof*(P[5]*This->m_lastField[0]-P[3]*This->m_lastField[2]) ; // dPy/ds
  dPdS[5] = This->m_cof*(P[3]*This->m_lastField[1]-P[4]*This->m_lastField[0]) ; // dPz/ds
}

/////////////////////////////////////////////////////////////////////////////////
// Inline methods
/////////////////////////////////////////////////////////////////////////////////
FQUALIFIER
void  GPNystromRK4_SetDistanceForConstantField(GPNystromRK4 *This, G4double length )
{
  This->m_magdistance=   length;
  This->m_magdistance2 = length*length;
}

FQUALIFIER
G4double  GPNystromRK4_GetDistanceForConstantField(GPNystromRK4 *This)
{
  return This->m_magdistance; 
}

/////////////////////////////////////////////////////////////////////////////////
// Get value of magnetic field while checking distance from last stored call
/////////////////////////////////////////////////////////////////////////////////

FQUALIFIER
void GPNystromRK4_getField (GPNystromRK4 *This, const G4double P[4])
{
  
  G4double dx = P[0]-This->m_fldPosition[0];
  G4double dy = P[1]-This->m_fldPosition[1];
  G4double dz = P[2]-This->m_fldPosition[2];

  if((dx*dx+dy*dy+dz*dz) > This->m_magdistance2)
  {
    This->m_fldPosition[0] = P[0];
    This->m_fldPosition[1] = P[1];
    This->m_fldPosition[2] = P[2];
    This->m_fldPosition[3] = P[3];   //  Generally it is P[7] - changed convention !!
    GPEquationOfMotion_GetFieldValue(This->m_fEq, This->m_fldPosition, This->m_lastField);
  }
}

//--------------------------------------------------------------
// class G4MagIntegratorStepper
//--------------------------------------------------------------  

// G4MagIntegratorStepper::G4MagIntegratorStepper
FQUALIFIER
void GPNystromRK4_G4MagIntegratorStepper_Constructor(GPNystromRK4 *This,
						     GPEquationOfMotion* Equation,
						     G4int       num_integration_vars,
						     G4int       num_state_vars)
{
  This->m_fEq = Equation;
  This->fNoIntegrationVariables = num_integration_vars;
  This->fNoStateVariables = num_state_vars;
}
