#include "hip/hip_runtime.h"
#include "GPOrb.h"
#include "GPUtils.h"

#define MAX(a,b) (((a)>(b))?(a):(b))

FQUALIFIER
void GPOrb_Constructor( GPOrb *This, G4double pRmax )
{
  This->fSolid.fType = kOrb;

  This->fRmax = pRmax;
  This->fRmaxTolerance = GPfmax( kRadTolerance, 2.e-11*This->fRmax);
}

FQUALIFIER
G4bool GPOrb_CalculateExtent(const GPOrb *This,
			     const EAxis pAxis,
			     GPVoxelLimits pVoxelLimit,
			     GPAffineTransform pTransform,
			     G4double* pMin, G4double* pMax)
{
    // Compute x/y/z mins and maxs for bounding box respecting limits,
    // with early returns if outside limits. Then switch() on pAxis,
    // and compute exact x and y limit for x/y case
      
    G4double xoffset,xMin,xMax;
    G4double yoffset,yMin,yMax;
    G4double zoffset,zMin,zMax;

    G4double diff1,diff2,maxDiff,newMin,newMax;
    G4double xoff1,xoff2,yoff1,yoff2;
    
    //    const G4double kCarTolerance = kCarTolerance;

    xoffset=GPAffineTransform_NetTranslation(&pTransform).x;
    xMin=xoffset-This->fRmax;
    xMax=xoffset+This->fRmax;

    if (GPVoxelLimits_IsXLimited(&pVoxelLimit))
    {
      if ( (xMin>GPVoxelLimits_GetMaxXExtent(&pVoxelLimit)+kCarTolerance)
        || (xMax<GPVoxelLimits_GetMinXExtent(&pVoxelLimit)-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (xMin<GPVoxelLimits_GetMinXExtent(&pVoxelLimit))
        {
          xMin=GPVoxelLimits_GetMinXExtent(&pVoxelLimit);
        }
        if (xMax>GPVoxelLimits_GetMaxXExtent(&pVoxelLimit))
        {
          xMax=GPVoxelLimits_GetMaxXExtent(&pVoxelLimit);
        }
      }
    }
    yoffset=GPAffineTransform_NetTranslation(&pTransform).y;
    yMin=yoffset-This->fRmax;
    yMax=yoffset+This->fRmax;

    if (GPVoxelLimits_IsYLimited(&pVoxelLimit))
    {
      if ( (yMin>GPVoxelLimits_GetMaxYExtent(&pVoxelLimit)+kCarTolerance)
        || (yMax<GPVoxelLimits_GetMinYExtent(&pVoxelLimit)-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (yMin<GPVoxelLimits_GetMinYExtent(&pVoxelLimit))
        {
          yMin=GPVoxelLimits_GetMinYExtent(&pVoxelLimit);
        }
        if (yMax>GPVoxelLimits_GetMaxYExtent(&pVoxelLimit))
        {
          yMax=GPVoxelLimits_GetMaxYExtent(&pVoxelLimit);
        }
      }
    }
    zoffset=GPAffineTransform_NetTranslation(&pTransform).z;
    zMin=zoffset-This->fRmax;
    zMax=zoffset+This->fRmax;

    if (GPVoxelLimits_IsZLimited(&pVoxelLimit))
    {
      if ( (zMin>GPVoxelLimits_GetMaxZExtent(&pVoxelLimit)+kCarTolerance)
        || (zMax<GPVoxelLimits_GetMinZExtent(&pVoxelLimit)-kCarTolerance) )
      {
        return false;
      }
      else
      {
        if (zMin<GPVoxelLimits_GetMinZExtent(&pVoxelLimit))
        {
          zMin=GPVoxelLimits_GetMinZExtent(&pVoxelLimit);
        }
        if (zMax>GPVoxelLimits_GetMaxZExtent(&pVoxelLimit))
        {
          zMax=GPVoxelLimits_GetMaxZExtent(&pVoxelLimit);
        }
      }
    }

    // Known to cut sphere

    switch (pAxis)
    {
      case kXAxis:
        yoff1=yoffset-yMin;
        yoff2=yMax-yoffset;

        if ( yoff1 >= 0 && yoff2 >= 0 )
        {
          // Y limits cross max/min x => no change
          //
          *pMin=xMin;
          *pMax=xMax;
        }
        else
        {
          // Y limits don't cross max/min x => compute max delta x,
          // hence new mins/maxs
          //
          diff1=sqrt(This->fRmax*This->fRmax-yoff1*yoff1);
          diff2=sqrt(This->fRmax*This->fRmax-yoff2*yoff2);
          maxDiff=(diff1>diff2) ? diff1:diff2;
          newMin=xoffset-maxDiff;
          newMax=xoffset+maxDiff;
          *pMin=(newMin<xMin) ? xMin : newMin;
          *pMax=(newMax>xMax) ? xMax : newMax;
        }
        break;
      case kYAxis:
        xoff1=xoffset-xMin;
        xoff2=xMax-xoffset;
        if (xoff1>=0&&xoff2>=0)
        {
          // X limits cross max/min y => no change
          //
          *pMin=yMin;
          *pMax=yMax;
        }
        else
        {
          // X limits don't cross max/min y => compute max delta y,
          // hence new mins/maxs
          //
          diff1=sqrt(This->fRmax*This->fRmax-xoff1*xoff1);
          diff2=sqrt(This->fRmax*This->fRmax-xoff2*xoff2);
          maxDiff=(diff1>diff2) ? diff1:diff2;
          newMin=yoffset-maxDiff;
          newMax=yoffset+maxDiff;
          *pMin=(newMin<yMin) ? yMin : newMin;
          *pMax=(newMax>yMax) ? yMax : newMax;
        }
        break;
      case kZAxis:
        *pMin=zMin;
        *pMax=zMax;
        break;
      default:
        break;
    }
    *pMin -= This->fRmaxTolerance;
    *pMax += This->fRmaxTolerance;

    return true;  
  
}

// --------------------------------------------------------------------

/////////////////////////////////////////////////////////////////////////
//
// Return whether point inside/outside/on surface, using tolerance

FQUALIFIER
EInside GPOrb_Inside( GEOMETRYLOC const GPOrb *This, 
		      GPThreeVector p)
{
  G4double rad2,tolRMax;
  EInside in;


  rad2 = GPThreeVector_mag2(p); //p.x*p.x+p.y*p.y+p.z*p.z ;

  G4double rad = sqrt(rad2);

  // G4double rad = std::sqrt(rad2);
  // Check radial surface
  // sets `in'
  
  tolRMax = This->fRmax - This->fRmaxTolerance*0.5 ;
    
  if ( rad <= tolRMax )  { in = kInside ; }
  else
  {
    tolRMax = This->fRmax + This->fRmaxTolerance*0.5 ;       
    if ( rad <= tolRMax )  { in = kSurface ; }
    else                   { in = kOutside ; }
  }
  return in;
}

///////////////////////////////////////////////////////////////////////
//
// Calculate side nearest to p, and return normal
// If two sides are equidistant, normal of first side (x/y/z) 
// encountered returned

FQUALIFIER
GPThreeVector GPOrb_SurfaceNormal( GEOMETRYLOC const GPOrb *This, 
				   GPThreeVector p)
{
  (void)This;
  /*ENorm side = kNRMax;
  GPThreeVector norm;
  G4double rad = sqrt(p.x*p.x+p.y*p.y+p.z*p.z);

  switch (side)
  {
    case kNRMax: 
      norm = GPThreeVector_create(p.x/rad,p.y/rad,p.z/rad);
      break;
   default:
      break;    
  } 

  return norm;*/
  return GPThreeVector_unit(p);
}



///////////////////////////////////////////////////////////////////////////
//
// Calculate distance to box from an outside point
// - return kInfinity if no intersection.
//
// ALGORITHM:
//
// Check that if point lies outside x/y/z extent of box, travel is towards
// the box (ie. there is a possibility of an intersection)
//
// Calculate pairs of minimum and maximum distances for x/y/z travel for
// intersection with the box's x/y/z extent.
// If there is a valid intersection, it is given by the maximum min distance
// (ie. distance to satisfy x/y/z intersections) *if* <= minimum max distance
// (ie. distance after which 1+ of x/y/z intersections not satisfied)
//
// NOTE:
//
// `Inside' safe - meaningful answers given if point is inside the exact
// shape.

FQUALIFIER
G4double GPOrb_DistanceToIn2( GEOMETRYLOC const GPOrb *This, 
			      GPThreeVector p,
			      GPThreeVector v)
{
  G4double snxt = kInfinity ;      // snxt = default return value

  G4double rad2, pDotV3d;
  G4double c, d2, s = kInfinity ;

  //const G4double dRmax = 100.*This->fRmax;

  // General Precalcs

  rad2    = GPThreeVector_mag2(p);// p.x*p.x + p.y*p.y + p.z*p.z ;
  pDotV3d = GPThreeVector_dot(p,v); //p.x*v.x + p.y*v.y + p.z*v.z ;

  // Radial Precalcs

  //tolORMax2 = (This->fRmax+This->fRmaxTolerance*0.5)*(This->fRmax+This->fRmaxTolerance*0.5) ;
  //tolIRMax2 = (This->fRmax-This->fRmaxTolerance*0.5)*(This->fRmax-This->fRmaxTolerance*0.5) ;

  // Outer spherical shell intersection
  // - Only if outside tolerant fRmax
  // - Check for if inside and outer GPOrb heading through solid (-> 0)
  // - No intersect -> no intersection with GPOrb
  //
  // Shell eqn: x^2+y^2+z^2 = RSPH^2
  //
  // => (px+svx)^2+(py+svy)^2+(pz+svz)^2=R^2
  //
  // => (px^2+py^2+pz^2) +2s(pxvx+pyvy+pzvz)+s^2(vx^2+vy^2+vz^2)=R^2
  // =>      rad2        +2s(pDotV3d)       +s^2                =R^2
  //
  // => s=-pDotV3d+-std::sqrt(pDotV3d^2-(rad2-R^2))


  G4double rad = sqrt(rad2);
  c = (rad - This->fRmax)*(rad + This->fRmax);

  if ( c > This->fRmaxTolerance*This->fRmax )
  {
    // If outside tolerant boundary of outer GPOrb
    // [ should be std::sqrt(rad2) - fRmax > fRmaxTolerance*0.5 ]

    d2 = pDotV3d*pDotV3d - c ;

    if ( d2 >= 0 )
    {
      s = -pDotV3d - sqrt(d2) ;
      if ( s >= 0 )
      {
		// TODO: check and fix This
        /*if ( s>dRmax ) // Avoid rounding errors due to precision issues seen on
        {              // 64 bits systems. Split long distances and recompute
          G4double fTerm = s-fmod(s,dRmax);
          s = fTerm + DistanceToIn(p+fTerm*v,v);
        } */
        return snxt = s;
      }
    }
    else    // No intersection with GPOrb
    {
      return snxt = kInfinity;
    }
  }
  else
  {
    if ( c > -This->fRmaxTolerance*This->fRmax )  // on surface  
    {
      d2 = pDotV3d*pDotV3d - c ;             
      if ( (d2 < This->fRmaxTolerance*This->fRmax) || (pDotV3d >= 0) )
      {
        return snxt = kInfinity;
      }
      else
      {
        return snxt = 0.;
      }
    }
  }
  return snxt;
}

//////////////////////////////////////////////////////////////////////////
// 
// Appoximate distance to box.
// Returns largest perpendicular distance to the closest x/y/z sides of
// the box, which is the most fast estimation of the shortest distance to box
// - If inside return 0

FQUALIFIER
G4double GPOrb_DistanceToIn( GEOMETRYLOC const GPOrb *This, 
			     GPThreeVector p)
{
  G4double safe = 0.0,
           rad  = GPThreeVector_mag(p); //sqrt(p.x*p.x+p.y*p.y+p.z*p.z);
  safe = rad - This->fRmax;
  if( safe < 0 ) { safe = 0.; }
  return safe;
}

/////////////////////////////////////////////////////////////////////////
//
// Calcluate distance to surface of box from inside
// by calculating distances to box's x/y/z planes.
// Smallest distance is exact distance to exiting.
// - Eliminate one side of each pair by considering direction of v
// - when leaving a surface & v.close, return 0

FQUALIFIER
G4double GPOrb_DistanceToOut2( GEOMETRYLOC const GPOrb *This, 
			       GPThreeVector p,GPThreeVector v,
                               const G4bool calcNorm,
			       G4bool *validNorm,GPThreeVector *n)
{
 G4double snxt = kInfinity;     // ??? snxt is default return value
  enum {kNull,kRMax} side = kNull;
  
  G4double rad2,pDotV3d; 
  GPThreeVector ipoint;  //G4double xi,yi,zi; // Intersection point
 
  G4double c,d2;
                 
  rad2    = GPThreeVector_mag2(p); //p.x*p.x + p.y*p.y + p.z*p.z;
  pDotV3d = GPThreeVector_dot(p,v); //p.x*v.x + p.y*v.y + p.z*v.z;
    
  // Radial Intersection from GPOrb::DistanceToIn
  //
  // Outer spherical shell intersection
  // - Only if outside tolerant fRmax
  // - Check for if inside and outer GPOrb heading through solid (-> 0)
  // - No intersect -> no intersection with GPOrb
  //
  // Shell eqn: x^2+y^2+z^2=RSPH^2
  //
  // => (px+svx)^2+(py+svy)^2+(pz+svz)^2=R^2
  //
  // => (px^2+py^2+pz^2) +2s(pxvx+pyvy+pzvz)+s^2(vx^2+vy^2+vz^2)=R^2
  // =>      rad2        +2s(pDotV3d)       +s^2                =R^2
  //
  // => s=-pDotV3d+-std::sqrt(pDotV3d^2-(rad2-R^2))
  
  const G4double  Rmax_plus = This->fRmax + This->fRmaxTolerance*0.5;
  G4double rad = sqrt(rad2);

  if ( rad <= Rmax_plus )
  {
    c = (rad - This->fRmax)*(rad + This->fRmax);

    if ( c < This->fRmaxTolerance*This->fRmax ) 
    {
      // Within tolerant Outer radius 
      // 
      // The test is
      //     rad  - fRmax < 0.5*fRmaxTolerance
      // =>  rad  < fRmax + 0.5*kRadTol
      // =>  rad2 < (fRmax + 0.5*kRadTol)^2
      // =>  rad2 < fRmax^2 + 2.*0.5*fRmax*kRadTol + 0.25*kRadTol*kRadTol
      // =>  rad2 - fRmax^2    <~    fRmax*kRadTol 

      d2 = pDotV3d*pDotV3d - c;

      if( ( c > -This->fRmaxTolerance*This->fRmax) &&         // on tolerant surface
          ( ( pDotV3d >= 0 )   || ( d2 < 0 )) )   // leaving outside from Rmax 
                                                  // not re-entering
      {
        if(calcNorm)
        {
          *validNorm = true ;
          *n         = GPThreeVector_create(p.x/This->fRmax,p.y/This->fRmax,p.z/This->fRmax) ;
        }
        return snxt = 0;
      }
      else 
      {
        snxt = -pDotV3d + sqrt(d2);    // second root since inside Rmax
        side = kRMax ; 
      }
    }
  }
  else // p is outside ???
  {
  }
  if (calcNorm)    // Output switch operator
  {
    switch( side )
    {
      case kRMax:
		ipoint = GPThreeVector_saxpy(snxt,v,p);
		*n=GPThreeVector_mult(ipoint,1.0/This->fRmax);
        //*n=GPThreeVector_create(xi/This->fRmax,yi/This->fRmax,zi/This->fRmax);
        *validNorm=true;
        break;
      default:
        break;
    }
  }
  return snxt;
}

////////////////////////////////////////////////////////////////////////////
//
// Calculate exact shortest distance to any boundary from inside
// - If outside return 0

FQUALIFIER
G4double GPOrb_DistanceToOut( GEOMETRYLOC const GPOrb *This, 
			      GPThreeVector p )
{
   G4double safe=0.0,rad = GPThreeVector_mag(p); //sqrt(p.x*p.x+p.y*p.y+p.z*p.z);

  safe = This->fRmax - rad;
  if ( safe < 0. ) safe = 0.;
  return safe;
}

