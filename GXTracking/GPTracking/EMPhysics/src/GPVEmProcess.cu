#include "hip/hip_runtime.h"
#include "GPVEmProcess.h"
#include "GPVEmModel.h"

#include "GPPhysicalConstants.h"
#include "GPSystemOfUnits.h"
#include "GPPhysicsTable.h"
#include "GPVParticleChange.h"
#include "GPRandom.h"

FQUALIFIER
GPVEmProcess::GPVEmProcess(hiprandState* devStates,
			   int threadId)
{
  //hiprand
  fThreadId = threadId;
  fDevStates = devStates;

  currentModel = 0;
  theLambdaTable = 0;
  theDensityFactor = 0;
  theDensityIdx = 0;

  // Size of tables assuming spline
  minKinEnergy = 0.1*keV;
  maxKinEnergy = 10.0*TeV;
  nLambdaBins  = 77;
  lambdaFactor  = 0.8;
  polarAngleLimit = 0.0;

  //  fParticleChange = 0;
  currentMaterial = 0;

  mfpKinEnergy  = DBL_MAX;
  preStepKinEnergy = 0.0;
  preStepLambda = 0.0;
  fFactor = 1.0;

  //Process and model
  isInitialised = false;

}

FQUALIFIER
GPVEmProcess::~GPVEmProcess()
{
  ;
}

FQUALIFIER
void GPVEmProcess::Clear()
{
  preStepLambda = 0.0;
  mfpKinEnergy  = DBL_MAX;
}

FQUALIFIER 
void GPVEmProcess::InitialiseProcess( /* G4VEmModel* model */)
{
  if(!isInitialised) {
    //@@@ Process sepecific implementation
    //SetEmModel
    //    currentModel = model;
    isInitialised = true;
  }
}

FQUALIFIER
void GPVEmProcess::StartTracking(GXTrack* track)
{
  // reset parameters for the new track
  //  currentParticle = track->GetParticleDefinition();
  theNumberOfInteractionLengthLeft = -1.0;
  mfpKinEnergy = DBL_MAX; 

}

FQUALIFIER
G4double GPVEmProcess::PostStepGetPhysicalInteractionLength(
                             GXTrack* track,
                             GPMaterial* material,
                             G4double   previousStepSize,
                             GPForceCondition* condition)
{
  *condition = NotForced;
  G4double x = DBL_MAX;

  preStepKinEnergy = track->E; //track.GetKineticEnergy();
  DefineMaterial(material);
  //@@@  SelectModel(preStepKinEnergy, currentCoupleIndex);

  if(!currentModel->IsActive(preStepKinEnergy)) { return x; }
 
  // compute mean free path
  if(preStepKinEnergy < mfpKinEnergy) {
    //    if (integral) { ComputeIntegralLambda(preStepKinEnergy); }
    //    else { 
    preStepLambda = GetCurrentLambda(preStepKinEnergy); 
    //    }

    // zero cross section
    if(preStepLambda <= 0.0) { 
      theNumberOfInteractionLengthLeft = -1.0;
      currentInteractionLength = DBL_MAX;
    }
  }

  // non-zero cross section
  if(preStepLambda > 0.0) { 

    if (theNumberOfInteractionLengthLeft < 0.0) {

      // beggining of tracking (or just after DoIt of this process)
      ResetNumberOfInteractionLengthLeft();

    } else if(currentInteractionLength < DBL_MAX) {

      // subtract NumberOfInteractionLengthLeft using previous step
      theNumberOfInteractionLengthLeft -= previousStepSize/currentInteractionLength;
      //SubtractNumberOfInteractionLengthLeft(previousStepSize);
      if(theNumberOfInteractionLengthLeft < 0.) {
	theNumberOfInteractionLengthLeft = 0.0;
      }
    }

    // new mean free path and step limit for the next step
    currentInteractionLength = 1.0/preStepLambda;
    x = theNumberOfInteractionLengthLeft * currentInteractionLength;
  }
  return x;
}

FQUALIFIER
GPVParticleChange* GPVEmProcess::PostStepDoIt(GXTrack* track,
                                              GPMaterial* material)
{
  // In all cases clear number of interaction lengths
  theNumberOfInteractionLengthLeft = -1.0;
  mfpKinEnergy = DBL_MAX; 

  //@@@implement for particle change for a specific model
  //  fParticleChange.InitializeForPostStep(track);

  // Do not make anything if particle is stopped, the annihilation then
  // should be performed by the AtRestDoIt!

  //@@@  if (track.GetTrackStatus() == fStopButAlive) { return &fParticleChange; }

  G4double finalT = track->E;//track.GetKineticEnergy();

  // Integral approach
  /*
  if (integral) {
    G4double lx = GetLambda(finalT, currentCouple);
    if(preStepLambda<lx && 1 < verboseLevel) {
      //warning
    }

    if(preStepLambda*G4UniformRand() > lx) {
      ClearNumberOfInteractionLengthLeft();
      return &fParticleChange;
    }
  }
  */

  //@@@  SelectModel(finalT, currentCoupleIndex);
  if(!currentModel->IsActive(finalT)) { return &fParticleChange; }

  // sample secondaries
  //  secParticles.clear();
  //  currentModel->SampleSecondaries(&secParticles, currentCouple, 
  //				  track.GetDynamicParticle(),
  //				  (*theCuts)[currentCoupleIndex]);
  currentModel->SampleSecondaries(track, material,1.0*mm);

  // save secondaries
  //  G4int num = secParticles.size();
  G4int num =1; //@@@ = 2 for GammaConversion
  if(num > 0) {

    //    fParticleChange.SetNumberOfSecondaries(num);
    G4double edep = fParticleChange.GetLocalEnergyDeposit();
    /*
    for (G4int i=0; i<num; ++i) {
      if (secParticles[i]) {
        G4DynamicParticle* dp = secParticles[i];
        const G4ParticleDefinition* p = dp->GetParticleDefinition();
        G4double e = dp->GetKineticEnergy();
        G4bool good = true;
        if(applyCuts) {
	  if (p == theGamma) {
	    if (e < (*theCutsGamma)[currentCoupleIndex]) { good = false; }
	    
	  } else if (p == theElectron) {
	    if (e < (*theCutsElectron)[currentCoupleIndex]) { good = false; }
	    
	  } else if (p == thePositron) {
	    if (electron_mass_c2 < (*theCutsGamma)[currentCoupleIndex] &&
		e < (*theCutsPositron)[currentCoupleIndex]) {
	      good = false;
	      e += 2.0*electron_mass_c2;
	    }
	  }
	  // added secondary if it is good
        }
        if (good) { 
          G4Track* t = new G4Track(dp, track.GetGlobalTime(),track.GetPosition());
          t->SetTouchableHandle(track.GetTouchableHandle());
          t->SetWeight(weight);
          pParticleChange->AddSecondary(t); 
        } else {
	  delete dp;
	  edep += e;
	}
      } 
    }
    */
    //@@@ get a secondary electron from the model - conversion need to add e+
    GXTrack t = currentModel->GetSecondary();
    fParticleChange.AddSecondary(t);
      
    fParticleChange.ProposeLocalEnergyDeposit(edep);
  }
  
  if(0.0 == fParticleChange.GetProposedKineticEnergy() &&
     fAlive == fParticleChange.GetTrackStatus()) {
    //  if(particle->GetProcessManager()->GetAtRestProcessVector()->size() > 0)
    //       { fParticleChange.ProposeTrackStatus(fStopButAlive); }
    //  else { 
    fParticleChange.ProposeTrackStatus(fStopAndKill); 
    //}
  }

  return &fParticleChange;
}

FQUALIFIER
G4double 
GPVEmProcess::CrossSectionPerVolume(G4double kineticEnergy,
				    GPMaterial* material)
{
  // Cross section per atom is calculated
  DefineMaterial(material);
  G4double cross = 0.0;
  if(theLambdaTable) {
    //    cross = (*theDensityFactor)[currentCoupleIndex]*
    //      (((*theLambdaTable)[basedCoupleIndex])->Value(kineticEnergy));
    cross = 1.0*
      theLambdaTable->physicsVectors[1].Value(kineticEnergy);

  } else {
    //    SelectModel(kineticEnergy, currentCoupleIndex);
    cross = currentModel->CrossSectionPerVolume(currentMaterial,
						kineticEnergy);
  }

  if(cross < 0.0) { cross = 0.0; }
  return cross;
}

FQUALIFIER
G4double GPVEmProcess::GetMeanFreePath(GPMaterial* material,
				       G4double kineticEnergy,
				       GPForceCondition* condition)
{
  *condition = NotForced;
  return GPVEmProcess::MeanFreePath(material,kineticEnergy);
}

FQUALIFIER
G4double GPVEmProcess::MeanFreePath(GPMaterial* material,
				    G4double kineticEnergy)
{
  //  DefineMaterial(track.GetMaterialCutsCouple());
  DefineMaterial(material);
  preStepLambda = GetCurrentLambda(kineticEnergy);
  G4double x = DBL_MAX;
  if(0.0 < preStepLambda) { x = 1.0/preStepLambda; }
  return x;
}

FQUALIFIER
G4double 
GPVEmProcess::ComputeCrossSectionPerAtom(G4double kineticEnergy, 
					 G4double Z, G4double A, G4double cut)
{
  // SelectModel(kineticEnergy, currentCoupleIndex);
  G4double x = 0.0;
  if(currentModel) {
    x = currentModel->ComputeCrossSectionPerAtom(kineticEnergy,
						 Z,A,cut);
  }
  return x;
}

FQUALIFIER
GPElement* GPVEmProcess::GetCurrentElement()
{
  GPElement* elm = 0;
  if(currentModel) {elm = currentModel->GetCurrentElement(); }
  return elm;
}


FQUALIFIER
void GPVEmProcess::SetMinKinEnergy(G4double e)
{
  nLambdaBins = GPlrint(nLambdaBins*log(maxKinEnergy/e)
			/log(maxKinEnergy/minKinEnergy));
  minKinEnergy = e;
}

FQUALIFIER
void GPVEmProcess::SetMaxKinEnergy(G4double e)
{
  nLambdaBins = GPlrint(nLambdaBins*log(e/minKinEnergy)
			/log(maxKinEnergy/minKinEnergy));
  maxKinEnergy = e;
}

//move to util
FQUALIFIER 
int GPVEmProcess::GPlrint(double ad)
{
  return (ad>0) ? (G4int)(ad+.5) : (G4int)(ad-.5);
}

// inline

FQUALIFIER 
void GPVEmProcess::DefineMaterial(GPMaterial* material)
{
  currentMaterial = material;
  fFactor = 1.0;
  mfpKinEnergy = DBL_MAX;
}

FQUALIFIER G4double GPVEmProcess::GetLambdaFromTable(G4double e)
{
  //  return ((*theLambdaTable)[basedCoupleIndex])->Value(e);
  return theLambdaTable->physicsVectors[1].Value(e);
}

FQUALIFIER G4double GPVEmProcess::ComputeCurrentLambda(G4double e)
{
  // return currentModel->CrossSectionPerVolume(baseMaterial,currentParticle,
  //                                           e,(*theCuts)[currentCoupleIndex]);
  //@@@ define theCuts value 
  return currentModel->CrossSectionPerVolume(currentMaterial,e,1.0*mm);
}

FQUALIFIER 
G4double GPVEmProcess::GetCurrentLambda(G4double e)
{
  G4double x;
  if(theLambdaTable)   { x = GetLambdaFromTable(e); }
  else                 { x = ComputeCurrentLambda(e); }
  return fFactor*x;
}

FQUALIFIER
G4double GPVEmProcess::GetLambda(G4double kinEnergy, 
				 GPMaterial* material)
{
  DefineMaterial(material);
  return GetCurrentLambda(kinEnergy);
}

FQUALIFIER 
G4double GPVEmProcess::MinKinEnergy()
{
  return minKinEnergy;
}

FQUALIFIER 
G4double GPVEmProcess::MaxKinEnergy()
{
  return maxKinEnergy;
}

FQUALIFIER
void GPVEmProcess::SetPolarAngleLimit(G4double val)
{
  if(val < 0.0)     { polarAngleLimit = 0.0; }
  else if(val > pi) { polarAngleLimit = pi;  }
  else              { polarAngleLimit = val; }
}

FQUALIFIER 
G4double GPVEmProcess::PolarAngleLimit()
{
  return polarAngleLimit;
}

FQUALIFIER 
void GPVEmProcess::SetLambdaFactor(G4double val)
{
  if(val > 0.0 && val <= 1.0) { lambdaFactor = val; }
}

//---------------------------------------------------------------------------
//
// G4VProcess
//
//---------------------------------------------------------------------------

FQUALIFIER
void GPVEmProcess::ResetNumberOfInteractionLengthLeft()
{
  theNumberOfInteractionLengthLeft = -log(GPUniformRand(fDevStates,fThreadId));
  theInitialNumberOfInteractionLength = theNumberOfInteractionLengthLeft; 
}

FQUALIFIER
void GPVEmProcess::EndTracking()
{
  theNumberOfInteractionLengthLeft = -1.0;
  currentInteractionLength = -1.0;
  theInitialNumberOfInteractionLength=-1.0;
}
